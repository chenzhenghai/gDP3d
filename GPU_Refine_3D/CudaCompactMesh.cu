#include "hip/hip_runtime.h"
#include "CudaCompactMesh.h"
#include "CudaMesh.h"

void compactMesh(
	int& out_numofpoint,
	double*& out_pointlist,
	double*& out_weightlist,
	RealD& t_pointlist,
	RealD& t_weightlist,
	int& out_numoftriface,
	int*& out_trifacelist,
	double*& out_trifacecent,
	IntD& t_trifacelist,
	RealD& t_trifacecent,
	TriStatusD& t_tristatus,
	TetHandleD& t_tri2tetlist,
	int& out_numoftet,
	int& out_numoftet_indomain,
	int*& out_tetlist,
	tetstatus*& out_tetstatus,
	IntD& t_tetlist,
	TetStatusD& t_tetstatus
)
{
	IntD t_sizes, t_indices, t_list;
	RealD t_list1;
	TetStatusD t_list2;
	int numberofthreads, numberofblocks;

	out_numofpoint = t_pointlist.size() / 3;
	out_pointlist = new double[3 * out_numofpoint];
	out_weightlist = new double[out_numofpoint];
	hipMemcpy(out_pointlist, thrust::raw_pointer_cast(&t_pointlist[0]), 3 * out_numofpoint * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(out_weightlist, thrust::raw_pointer_cast(&t_weightlist[0]), out_numofpoint * sizeof(double), hipMemcpyDeviceToHost);

	int last_triface = t_tristatus.size();
	t_sizes.resize(last_triface);
	t_indices.resize(last_triface);
	thrust::fill(t_sizes.begin(), t_sizes.end(), 1);
	thrust::replace_if(t_sizes.begin(), t_sizes.end(), t_tristatus.begin(), isEmptyTri(), 0);
	thrust::exclusive_scan(t_sizes.begin(), t_sizes.end(), t_indices.begin());
	out_numoftriface = thrust::reduce(t_sizes.begin(), t_sizes.end());
	out_trifacelist = new int[3 * out_numoftriface];
	out_trifacecent = new double[3 * out_numoftriface];
	t_list.resize(3 * out_numoftriface);
	t_list1.resize(3 * out_numoftriface);

	numberofthreads = last_triface;
	numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);
	kernelCompactTriface << <numberofblocks, BLOCK_SIZE >> > (
		thrust::raw_pointer_cast(&t_trifacelist[0]),
		thrust::raw_pointer_cast(&t_trifacecent[0]),
		thrust::raw_pointer_cast(&t_tri2tetlist[0]),
		thrust::raw_pointer_cast(&t_tetstatus[0]),
		thrust::raw_pointer_cast(&t_sizes[0]),
		thrust::raw_pointer_cast(&t_indices[0]),
		thrust::raw_pointer_cast(&t_list[0]),
		thrust::raw_pointer_cast(&t_list1[0]),
		numberofthreads
		);
	hipMemcpy(out_trifacelist, thrust::raw_pointer_cast(&t_list[0]), 3 * out_numoftriface * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(out_trifacecent, thrust::raw_pointer_cast(&t_list1[0]), 3 * out_numoftriface * sizeof(double), hipMemcpyDeviceToHost);

	int last_tet = t_tetstatus.size();
	t_sizes.resize(last_tet);
	t_indices.resize(last_tet);
	thrust::fill(t_sizes.begin(), t_sizes.end(), 1);
	numberofthreads = last_tet;
	numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);

	kernelCompactTet_Phase1 << <numberofblocks, BLOCK_SIZE >> > (
		thrust::raw_pointer_cast(&t_tetlist[0]),
		thrust::raw_pointer_cast(&t_tetstatus[0]),
		thrust::raw_pointer_cast(&t_sizes[0]),
		numberofthreads
		);

	thrust::exclusive_scan(t_sizes.begin(), t_sizes.end(), t_indices.begin());
	out_numoftet = thrust::reduce(t_sizes.begin(), t_sizes.end());
	out_tetlist = new int[4 * out_numoftet];
	out_tetstatus = new tetstatus[out_numoftet];
	t_list.resize(4 * out_numoftet);
	t_list2.resize(out_numoftet);

	kernelCompactTet_Phase2 << <numberofblocks, BLOCK_SIZE >> > (
		thrust::raw_pointer_cast(&t_tetlist[0]),
		thrust::raw_pointer_cast(&t_tetstatus[0]),
		thrust::raw_pointer_cast(&t_sizes[0]),
		thrust::raw_pointer_cast(&t_indices[0]),
		thrust::raw_pointer_cast(&t_list[0]),
		thrust::raw_pointer_cast(&t_list2[0]),
		numberofthreads
		);
	hipMemcpy(out_tetlist, thrust::raw_pointer_cast(&t_list[0]), 4 * out_numoftet * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(out_tetstatus, thrust::raw_pointer_cast(&t_list2[0]), out_numoftet * sizeof(tetstatus), hipMemcpyDeviceToHost);

	int numoftets_indomain = 0;
	for (int i = 0; i < out_numoftet; i++)
	{
		if (out_tetstatus[i].isInDomain())
			numoftets_indomain++;
	}
	out_numoftet_indomain = numoftets_indomain;
}