#include "hip/hip_runtime.h"
#include "CudaInsertPoint.h"
#include "CudaMesh.h"
#include "CudaAnimation.h"
#include <hip/hip_math_constants.h>
#include <time.h>

int insertPoint_New(
	IntD& t_aabbnodeleft,
	IntD& t_aabbnoderight,
	RealD& t_aabbnodebbs,
	RealD& t_aabbpmcoord,
	RealD& t_aabbpmbbs,
	TetHandleD& t_recordoldtetlist,
	IntD& t_recordoldtetidx,
	RealD& t_pointlist,
	RealD& t_weightlist,
	PointTypeD& t_pointtypelist,
	IntD& t_pointpmt,
	IntD& t_trifacelist,
	RealD& t_trifacecent,
	TetHandleD& t_tri2tetlist,
	TriStatusD& t_tristatus,
	IntD& t_trifacepmt,
	IntD& t_tetlist,
	TetHandleD& t_neighborlist,
	TriHandleD& t_tet2trilist,
	TetStatusD& t_tetstatus,
	IntD& t_insertidxlist,
	IntD& t_threadmarker,
	int numofbadelements,
	int numofbadtriface,
	int numofbadtet,
	int& numofpoints,
	int& numoftriface,
	int& numoftet,
	MESHCR* criteria,
	MESHIO* inputmesh,
	MESHBH* behavior,
	int insertmode, // 0: subface only, 1: mix subface and tet
	int iter
)
{
#ifdef GQM3D_CHECKMEMORY
	hipDeviceSynchronize();
	gpuMemoryCheck();
#endif

#ifdef GQM3D_PROFILING
	clock_t tv[2];
	hipDeviceSynchronize();
	tv[0] = clock();
#endif

#ifdef GQM3D_LOOP_PROFILING
	clock_t ltv[2];
#endif

	internalmesh* drawmesh = behavior->drawmesh;
	int numofinsertpt = numofbadelements;
	REAL aabb_diglen = inputmesh->aabb_diglen;
	int aabb_level = inputmesh->aabb_level;
	int aabb_pmnum = inputmesh->numofaabbpms;

	// Initialization
	int numberofwonfacets;
	int numberofnewtets_facet;
	int numberofthreads;
	int numberofblocks;
	IntD t_threadlist; // active thread list
	UInt64D t_tetmarker(numoftet, 0); // marker for tets. Used for cavity.

	RealD t_insertptlist(3 * numofinsertpt);
	IntD t_priority(numofinsertpt, 0);
	RealD t_priorityreal(numofinsertpt, 0.0); // store real temporarily

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        vector initialization time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["vector initialization"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	// Compute Steiner points and priorities
	numberofblocks = (ceil)((float)numofinsertpt / BLOCK_SIZE);
	kernelComputePriorities << <numberofblocks, BLOCK_SIZE >> > (
		thrust::raw_pointer_cast(&t_pointlist[0]),
		thrust::raw_pointer_cast(&t_trifacelist[0]),
		thrust::raw_pointer_cast(&t_tetlist[0]),
		thrust::raw_pointer_cast(&t_insertidxlist[0]),
		thrust::raw_pointer_cast(&t_threadmarker[0]),
		thrust::raw_pointer_cast(&t_priorityreal[0]),
		numofinsertpt
		);

#ifdef GQM3D_DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        kernelComputePriorities time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["kernelComputePriorities"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	// Sort element indices by priorites and
	// pick the first N elements where N = behavior->maxbadelements
	if (behavior->filtermode == 2 && behavior->maxbadelements > 0 && numofbadelements > behavior->maxbadelements)
	{
		if (behavior->filterstatus == 1)
			behavior->filterstatus = 2;

		int numberofloser;
		if (numofbadtriface > numofbadtet)
		{
			numberofloser = numofbadtriface - behavior->maxbadelements;
			if (numberofloser > 0)
			{
				thrust::sort_by_key(t_insertidxlist.begin(), t_insertidxlist.begin() + numofbadtriface,
					t_priorityreal.begin());
				thrust::fill(t_threadmarker.begin(), t_threadmarker.begin() + numberofloser, -1);
			}
		}
		else
		{
			numberofloser = numofbadtet - behavior->maxbadelements;
			if (numberofloser > 0)
			{
				thrust::sort_by_key(t_insertidxlist.begin() + numofbadtriface, t_insertidxlist.end(),
					t_priorityreal.begin() + numofbadtriface);
				thrust::fill(t_threadmarker.begin() + numofbadtriface, t_threadmarker.begin() + numofbadtriface + numberofloser, -1);
			}
		}

#ifdef GQM3D_PROFILING
		hipDeviceSynchronize();
		tv[1] = clock();
		printf("        fast filtering - sorting time = %f\n", (REAL)(tv[1] - tv[0]));
		inserttimer["fast filtering - sorting"] += (REAL)(tv[1] - tv[0]);
		tv[0] = tv[1];
#endif
	}
	else
	{
		if (behavior->filterstatus == 2)
			behavior->filterstatus = 3;
	}

	kernelComputeSteinerPoints << <numberofblocks, BLOCK_SIZE >> > (
		thrust::raw_pointer_cast(&t_pointlist[0]),
		thrust::raw_pointer_cast(&t_weightlist[0]),
		thrust::raw_pointer_cast(&t_trifacelist[0]),
		thrust::raw_pointer_cast(&t_trifacecent[0]),
		thrust::raw_pointer_cast(&t_tristatus[0]),
		thrust::raw_pointer_cast(&t_tetlist[0]),
		thrust::raw_pointer_cast(&t_tetstatus[0]),
		thrust::raw_pointer_cast(&t_insertidxlist[0]),
		thrust::raw_pointer_cast(&t_threadmarker[0]),
		thrust::raw_pointer_cast(&t_insertptlist[0]),
		numofinsertpt
		);

#ifdef GQM3D_DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        kernelComputeSteinerPoints time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["kernelComputeSteinerPoints"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	// Modify priorities and convert them into integers
	// Make sure triface > tet
	double priority_min[2], priority_max[2], priority_offset[2] = { 0, 0 };
	thrust::pair<RealD::iterator, RealD::iterator> priority_pair;
	if (numofbadtet > 0)
	{
		priority_pair =
			thrust::minmax_element(
				t_priorityreal.begin() + numofbadtriface,
				t_priorityreal.end());
		priority_min[1] = *priority_pair.first;
		priority_max[1] = *priority_pair.second;
		priority_offset[1] = 0;
#ifdef GQM3D_DEBUG
		printf("MinMax Real priorities for tet: %lf, %lf\n", priority_min[1], priority_max[1]);
		printf("Offset: %lf\n", priority_offset[1]);
#endif
	}

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        min-max tet priority time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["min-max tet priority"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	if (numofbadtriface > 0)
	{
		priority_pair =
			thrust::minmax_element(
				t_priorityreal.begin(),
				t_priorityreal.begin() + numofbadtriface);
		priority_min[0] = *priority_pair.first;
		priority_max[0] = *priority_pair.second;
		if (numofbadtet > 0)
			priority_offset[0] = priority_max[1] + priority_offset[1] + 10 - priority_min[0];
		else
			priority_offset[0] = 0;
#ifdef GQM3D_DEBUG
		printf("MinMax Real priorities for subface: %lf, %lf\n", priority_min[0], priority_max[0]);
		printf("Offset: %lf\n", priority_offset[0]);
#endif
	}

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        min-max subface priority time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["min-max subface priority"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	kernelModifyPriority << <numberofblocks, BLOCK_SIZE >> > (
		thrust::raw_pointer_cast(&t_priorityreal[0]),
		thrust::raw_pointer_cast(&t_priority[0]),
		priority_offset[0],
		priority_offset[1],
		thrust::raw_pointer_cast(&t_tristatus[0]),
		thrust::raw_pointer_cast(&t_tetstatus[0]),
		thrust::raw_pointer_cast(&t_insertidxlist[0]),
		thrust::raw_pointer_cast(&t_threadmarker[0]),
		numofbadtriface,
		numofinsertpt
		);

#ifdef GQM3D_DEBUG
	if (numofbadtriface > 0)
	{
		priority_pair =
			thrust::minmax_element(
				t_priorityreal.begin(),
				t_priorityreal.begin() + numofbadtriface);
		priority_min[0] = *priority_pair.first;
		priority_max[0] = *priority_pair.second;
		printf("MinMax Real priorities for subface: %lf, %lf\n", priority_min[0], priority_max[0]);
	}

	if (numofbadtet > 0)
	{
		priority_pair =
			thrust::minmax_element(
				t_priorityreal.begin() + numofbadtriface,
				t_priorityreal.end());
		priority_min[1] = *priority_pair.first;
		priority_max[1] = *priority_pair.second;
		printf("MinMax Real priorities for tet: %lf, %lf\n", priority_min[1], priority_max[1]);
	}

	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        kernelModifyPriority time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["kernelModifyPriority"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	freeVec(t_priorityreal);

	if (behavior->filtermode == 2 && behavior->maxbadelements > 0 && numofbadelements > behavior->maxbadelements)
	{
		int gridlength = 150; // ^3
		int totalgridsize = gridlength * gridlength * gridlength;
		if (t_tetmarker.size() < totalgridsize)
			t_tetmarker.resize(totalgridsize, 0);

		int range_left, range_right;
		if (numofbadtriface > numofbadtet)
		{
			range_left = 0;
			range_right = numofbadtriface;
		}
		else
		{
			range_left = numofbadtriface;
			range_right = numofbadelements;
		}

		double step_x = (inputmesh->aabb_xmax - inputmesh->aabb_xmin) / gridlength;
		double step_y = (inputmesh->aabb_ymax - inputmesh->aabb_ymin) / gridlength;
		double step_z = (inputmesh->aabb_zmax - inputmesh->aabb_zmin) / gridlength;

		kernelGridFiltering << <numberofblocks, BLOCK_SIZE >> > (
			thrust::raw_pointer_cast(&t_priority[0]),
			thrust::raw_pointer_cast(&t_tetmarker[0]),
			thrust::raw_pointer_cast(&t_insertidxlist[0]),
			thrust::raw_pointer_cast(&t_threadmarker[0]),
			thrust::raw_pointer_cast(&t_insertptlist[0]),
			range_left,
			range_right,
			step_x,
			step_y,
			step_z,
			inputmesh->aabb_xmin,
			inputmesh->aabb_ymin,
			inputmesh->aabb_zmin,
			gridlength
			);

		thrust::fill(t_tetmarker.begin(), t_tetmarker.begin() + totalgridsize, 0);
#ifdef GQM3D_PROFILING
		hipDeviceSynchronize();
		tv[1] = clock();
		printf("        fast filtering - grid time = %f\n", (REAL)(tv[1] - tv[0]));
		inserttimer["fast filtering - grid"] += (REAL)(tv[1] - tv[0]);
		tv[0] = tv[1];
#endif
	}

	// Update working thread list
	numberofthreads = updateActiveListByMarker(t_threadmarker, t_threadlist, t_threadmarker.size());
	numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);
	if (numberofthreads == 0)
	{
		if (behavior->R5) // no more bad elements
			return 0;
		else
			return 1;
	}

	if (behavior->verbose >= 1)
		printf("        After Steiner point and priority, numberofthreads = %d(#%d, #%d, #%d)\n",
			numberofthreads,
			thrust::count(t_threadmarker.begin(), t_threadmarker.end(), 0),
			thrust::count(t_threadmarker.begin(), t_threadmarker.end(), 1),
			thrust::count(t_threadmarker.begin(), t_threadmarker.end(), 2));

#ifdef GQM3D_CHECKMEMORY
	hipDeviceSynchronize();
	gpuMemoryCheck();
#endif

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        update working thread list time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["update working thread list"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	// Locate Steiner points
	thrust::device_vector<locateresult> t_pointlocation(numofinsertpt, UNKNOWN);
	TetHandleD t_searchtet(numofinsertpt, tethandle(-1, 11));

	kernelLocatePoint << <numberofblocks, BLOCK_SIZE >> > (
		thrust::raw_pointer_cast(&t_pointlist[0]),
		thrust::raw_pointer_cast(&t_weightlist[0]),
		thrust::raw_pointer_cast(&t_trifacelist[0]),
		thrust::raw_pointer_cast(&t_tri2tetlist[0]),
		thrust::raw_pointer_cast(&t_tristatus[0]),
		thrust::raw_pointer_cast(&t_tetlist[0]),
		thrust::raw_pointer_cast(&t_neighborlist[0]),
		thrust::raw_pointer_cast(&t_tetstatus[0]),
		thrust::raw_pointer_cast(&t_priority[0]),
		thrust::raw_pointer_cast(&t_pointlocation[0]),
		thrust::raw_pointer_cast(&t_searchtet[0]),
		thrust::raw_pointer_cast(&t_insertidxlist[0]),
		thrust::raw_pointer_cast(&t_threadmarker[0]),
		thrust::raw_pointer_cast(&t_threadlist[0]),
		thrust::raw_pointer_cast(&t_insertptlist[0]),
		numberofthreads
		);

#ifdef GQM3D_DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        kernelLocatePoint time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["kernelLocatePoint"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	// update working thread list
	numberofthreads = updateActiveListByMarker(t_threadmarker, t_threadlist, t_threadmarker.size());
	numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);
	if (behavior->verbose >= 1)
		printf("        After point location, numberofthreads = %d(#%d, #%d, #%d)\n",
			numberofthreads,
			thrust::count(t_threadmarker.begin(), t_threadmarker.end(), 0),
			thrust::count(t_threadmarker.begin(), t_threadmarker.end(), 1),
			thrust::count(t_threadmarker.begin(), t_threadmarker.end(), 2));
	if (numberofthreads == 0)
		return 0;

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        update working thread list time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["update working thread list"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	if (drawmesh != NULL && drawmesh->animation)
	{
		if (iter == drawmesh->iter_tet)
			outputStartingFrame(
				drawmesh,
				t_pointlist,
				t_tetlist,
				t_tetstatus,
				t_threadlist,
				t_insertidxlist,
				t_insertptlist,
				t_searchtet,
				-1,
				-1,
				iter
			);
	}

#ifdef GQM3D_CHECKMEMORY
	//hipDeviceSetLimit(hipLimitStackSize, 0); // free memory used by kernel
	hipDeviceSynchronize();
	gpuMemoryCheck();
#endif

	TetHandleD t_caveoldtetlist; // list to record interior tets
	IntD t_caveoldtetidx;
	TetHandleD t_cavetetlist; // list to record tets in expanding cavities
	IntD t_cavetetidx;
	TetHandleD t_cavebdrylist; // list to record boundary tets
	IntD t_cavebdryidx;

	// Adatively reserve memory space
	//if (!behavior->R5)
	{
		// size and fac would fluctuate
		if (behavior->caveoldtetsizefac > 3.0)
			behavior->caveoldtetsizefac = 1.5;
		if (behavior->cavetetsizefac > 3.0)
			behavior->cavetetsizefac = 1.5;
		if (behavior->cavebdrysizefac > 3.0)
			behavior->cavebdrysizefac = 1.5;
	}
	//printf("behavior->caveoldtetsize = %d, behavior->caveoldtetsizefac = %lf\n", behavior->caveoldtetsize, behavior->caveoldtetsizefac);
	int resoldtetsize = behavior->caveoldtetsize * behavior->caveoldtetsizefac;
	t_caveoldtetlist.reserve(resoldtetsize);
	t_caveoldtetidx.reserve(resoldtetsize);
	//printf("behavior->cavetetsize = %d, behavior->cavetetsizefac = %lf\n", behavior->cavetetsize, behavior->cavetetsizefac);
	int restetsize = behavior->cavetetsize * behavior->cavetetsizefac;
	t_cavetetlist.reserve(restetsize);
	t_cavetetidx.reserve(restetsize);
	//printf("behavior->cavebdrysize = %d, behavior->cavebdrysizefac = %lf\n", behavior->cavebdrysize, behavior->cavebdrysizefac);
	int resbdrysize = behavior->cavebdrysize * behavior->cavebdrysizefac;
	t_cavebdrylist.reserve(resbdrysize);
	t_cavebdryidx.reserve(resbdrysize);

	// Compute initial cavity starting points
	int oldsize, newsize;
	int initialcavitysize/*, initialsubcavitysize*/;
	IntD t_initialcavitysize(numofinsertpt, MAXINT);
	IntD t_initialcavityindices(numofinsertpt, -1);

	// set losers' cavity and scavity sizes to zero
	thrust::replace_if(t_initialcavitysize.begin(), t_initialcavitysize.end(), t_threadmarker.begin(), isNegativeInt(), 0);

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        cavity vector initialization time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["cavity vector initialization"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	// Form initial cavities
	// mark and count the initial cavities
	// mark tets using original thread indices
	kernelMarkAndCountInitialCavity << <numberofblocks, BLOCK_SIZE >> > (
		thrust::raw_pointer_cast(&t_insertidxlist[0]),
		thrust::raw_pointer_cast(&t_pointlocation[0]),
		thrust::raw_pointer_cast(&t_threadlist[0]),
		thrust::raw_pointer_cast(&t_searchtet[0]),
		thrust::raw_pointer_cast(&t_neighborlist[0]),
		thrust::raw_pointer_cast(&t_tristatus[0]),
		thrust::raw_pointer_cast(&t_tetstatus[0]),
		thrust::raw_pointer_cast(&t_priority[0]),
		thrust::raw_pointer_cast(&t_tetmarker[0]),
		thrust::raw_pointer_cast(&t_threadmarker[0]),
		thrust::raw_pointer_cast(&t_initialcavitysize[0]),
		numberofthreads
		);

#ifdef GQM3D_DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        kernelMarkAndCountInitialCavity time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["kernelMarkAndCountInitialCavity"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	// Check record oldtet lists
	if (behavior->cavitymode == 2)
	{
		numberofthreads = t_recordoldtetidx.size();
		//printf("t_recordoldtet size = %d\n", numberofthreads);
		if (numberofthreads > 0)
		{
			numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);
			kernelCheckRecordOldtet << <numberofblocks, BLOCK_SIZE >> > (
				thrust::raw_pointer_cast(&t_recordoldtetlist[0]),
				thrust::raw_pointer_cast(&t_recordoldtetidx[0]),
				thrust::raw_pointer_cast(&t_insertidxlist[0]),
				thrust::raw_pointer_cast(&t_insertptlist[0]),
				thrust::raw_pointer_cast(&t_pointlist[0]),
				thrust::raw_pointer_cast(&t_weightlist[0]),
				thrust::raw_pointer_cast(&t_tetlist[0]),
				thrust::raw_pointer_cast(&t_neighborlist[0]),
				thrust::raw_pointer_cast(&t_tristatus[0]),
				thrust::raw_pointer_cast(&t_tetstatus[0]),
				thrust::raw_pointer_cast(&t_priority[0]),
				thrust::raw_pointer_cast(&t_tetmarker[0]),
				thrust::raw_pointer_cast(&t_threadmarker[0]),
				thrust::raw_pointer_cast(&t_initialcavitysize[0]),
				numofbadtriface,
				numofbadelements,
				numberofthreads
				);

#ifdef GQM3D_DEBUG
			gpuErrchk(hipPeekAtLastError());
			gpuErrchk(hipDeviceSynchronize());
#endif

			kernelKeepRecordOldtet << <numberofblocks, BLOCK_SIZE >> > (
				thrust::raw_pointer_cast(&t_recordoldtetidx[0]),
				thrust::raw_pointer_cast(&t_insertidxlist[0]),
				thrust::raw_pointer_cast(&t_threadmarker[0]),
				numberofthreads
				);

#ifdef GQM3D_DEBUG
			gpuErrchk(hipPeekAtLastError());
			gpuErrchk(hipDeviceSynchronize());
#endif
		}
	}

	// update working thread list
	numberofthreads = updateActiveListByMarker(t_threadmarker, t_threadlist, t_threadmarker.size());
	if (behavior->verbose >= 1)
		printf("        After initial cavity marking, numberofthreads = %d(#%d, #%d, #%d)\n",
			numberofthreads,
			thrust::count(t_threadmarker.begin(), t_threadmarker.end(), 0),
			thrust::count(t_threadmarker.begin(), t_threadmarker.end(), 1),
			thrust::count(t_threadmarker.begin(), t_threadmarker.end(), 2));
	if (numberofthreads == 0)
	{
		// This should not error
		printf("Error: 0 threads after marking initial cavities!\n");
		exit(0);
	}

#ifdef GQM3D_CHECKMEMORY
	hipDeviceSynchronize();
	gpuMemoryCheck();
#endif

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        update working thread list time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["update working thread list"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	// compute total size and indices for intital cavities
	thrust::exclusive_scan(t_initialcavitysize.begin(), t_initialcavitysize.end(), t_initialcavityindices.begin());
	initialcavitysize = t_initialcavityindices[numofinsertpt - 1] + t_initialcavitysize[numofinsertpt - 1];

#ifdef GQM3D_DEBUG
	printf("Initial cavity size = %d\n", initialcavitysize);
#endif

	// init cavity lists
	numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);
	t_caveoldtetlist.resize(initialcavitysize);
	t_caveoldtetidx.resize(initialcavitysize);

	int expandfactor = 4;
	t_cavetetlist.resize(expandfactor * initialcavitysize);
	t_cavetetidx.resize(expandfactor * initialcavitysize);
	thrust::fill(t_cavetetidx.begin(), t_cavetetidx.end(), -1);

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        prepare vector for intial cavity time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["prepare vector for intial cavity"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	kernelInitCavityLinklist << < numberofblocks, BLOCK_SIZE >> > (
		thrust::raw_pointer_cast(&t_insertidxlist[0]),
		thrust::raw_pointer_cast(&t_pointlocation[0]),
		thrust::raw_pointer_cast(&t_threadlist[0]),
		thrust::raw_pointer_cast(&t_searchtet[0]),
		thrust::raw_pointer_cast(&t_neighborlist[0]),
		thrust::raw_pointer_cast(&t_tristatus[0]),
		thrust::raw_pointer_cast(&t_tetstatus[0]),
		thrust::raw_pointer_cast(&t_initialcavityindices[0]),
		thrust::raw_pointer_cast(&t_caveoldtetlist[0]),
		thrust::raw_pointer_cast(&t_caveoldtetidx[0]),
		thrust::raw_pointer_cast(&t_cavetetlist[0]),
		thrust::raw_pointer_cast(&t_cavetetidx[0]),
		thrust::raw_pointer_cast(&t_threadmarker[0]),
		numberofthreads
		);

#ifdef GQM3D_DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

	freeVec(t_pointlocation);
	freeVec(t_initialcavitysize);
	freeVec(t_initialcavityindices);

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        kernelInitCavityLinklist = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["kernelInitCavityLinklist"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	if (behavior->cavitymode == 2)
	{
		auto first_record_iter = thrust::make_zip_iterator(thrust::make_tuple(t_recordoldtetlist.begin(), t_recordoldtetidx.begin()));
		auto last_record_iter = thrust::make_zip_iterator(thrust::make_tuple(t_recordoldtetlist.end(), t_recordoldtetidx.end()));

		int expandreusesize = thrust::count_if(t_recordoldtetidx.begin(), t_recordoldtetidx.end(), isTetIndexToReuse());
		//printf("expandreusesize = %d\n", expandreusesize);

		if (expandreusesize > 0)
		{
			// copy recordoldtet to oldtet
			int oldlistsize = t_caveoldtetlist.size();
			t_caveoldtetlist.resize(oldlistsize + expandreusesize);
			t_caveoldtetidx.resize(oldlistsize + expandreusesize);
			auto first_old_iter =
				thrust::make_zip_iterator(thrust::make_tuple(t_caveoldtetlist.begin() + oldlistsize, t_caveoldtetidx.begin() + oldlistsize));
			auto last_old_iter =
				thrust::copy_if(first_record_iter, last_record_iter, first_old_iter, isCavityTupleToReuse());
			//printf("distance = %d\n", thrust::distance(first_old_iter, last_old_iter));

			numberofthreads = expandreusesize; // each thread works on one tet in cavetetlist
			numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);

			kernelSetReuseOldtet << < numberofblocks, BLOCK_SIZE >> > (
				thrust::raw_pointer_cast(&t_caveoldtetlist[0]),
				thrust::raw_pointer_cast(&t_caveoldtetidx[0]),
				oldlistsize,
				numberofthreads
				);

#ifdef GQM3D_DEBUG
			gpuErrchk(hipPeekAtLastError());
			gpuErrchk(hipDeviceSynchronize());
#endif

			// expand cavetet
			IntD t_cavetetexpandsize(numberofthreads, 0), t_cavetetexpandindices(numberofthreads, -1);
			int cavetetexpandsize;

			kernelCheckCavetetFromReuseOldtet << < numberofblocks, BLOCK_SIZE >> >(
				thrust::raw_pointer_cast(&t_caveoldtetlist[0]),
				thrust::raw_pointer_cast(&t_caveoldtetidx[0]),
				thrust::raw_pointer_cast(&t_neighborlist[0]),
				thrust::raw_pointer_cast(&t_cavetetexpandsize[0]),
				thrust::raw_pointer_cast(&t_tetmarker[0]),
				oldlistsize,
				numberofthreads
				);

			thrust::exclusive_scan(t_cavetetexpandsize.begin(), t_cavetetexpandsize.end(), t_cavetetexpandindices.begin());
			cavetetexpandsize = t_cavetetexpandindices[numberofthreads - 1] + t_cavetetexpandsize[numberofthreads - 1];
			//printf("cavetetexpandsize = %d\n", cavetetexpandsize);
			int oldcavetetsize = t_cavetetlist.size();
			t_cavetetlist.resize(oldcavetetsize + cavetetexpandsize);
			t_cavetetidx.resize(oldcavetetsize + cavetetexpandsize);

			kernelAppendCavetetFromReuseOldtet << < numberofblocks, BLOCK_SIZE >> >(
				thrust::raw_pointer_cast(&t_caveoldtetlist[0]),
				thrust::raw_pointer_cast(&t_caveoldtetidx[0]),
				thrust::raw_pointer_cast(&t_cavetetlist[0]),
				thrust::raw_pointer_cast(&t_cavetetidx[0]),
				thrust::raw_pointer_cast(&t_neighborlist[0]),
				thrust::raw_pointer_cast(&t_cavetetexpandindices[0]),
				thrust::raw_pointer_cast(&t_tetmarker[0]),
				oldlistsize,
				oldcavetetsize,
				numberofthreads
				);
		}

		// remove used recordoldtet
		//printf("before remove: t_recordoldtet size = %d\n", t_recordoldtetlist.size());
		auto last_record_iter_remove = thrust::remove_if(first_record_iter, last_record_iter, isInvalidCavityTuple());
		int newlistsize = thrust::distance(first_record_iter, last_record_iter_remove);
		t_recordoldtetlist.resize(newlistsize);
		t_recordoldtetidx.resize(newlistsize);
		//printf("After remove: t_recordoldtet size = %d\n", t_recordoldtetlist.size());

	}

#ifdef GQM3D_DEBUG
	//{
	//	bool error = false;
	//	tethandle* tmptetlist = new tethandle[t_caveoldtetlist.size()];
	//	thrust::copy(t_caveoldtetlist.begin(), t_caveoldtetlist.end(), tmptetlist);
	//	int* tmpidxlist = new int[t_caveoldtetidx.size()];
	//	thrust::copy(t_caveoldtetidx.begin(), t_caveoldtetidx.end(), tmpidxlist);

	//	printf("after initial cavity caveoldtetlist:\n");
	//	for (int i = 0; i < t_caveoldtetlist.size(); i++)
	//	{
	//		tethandle tmp = tmptetlist[i];
	//		int tmpidx = tmpidxlist[i];
	//		//printf("%d, %d, %d\n", tmp.id, tmp.ver, tmpidx);
	//		if (tmp.id < 0)
	//		{
	//			printf("tet.id = %d, threadId = %d\n", tmp.id, tmpidx);
	//			error = true;
	//			break;
	//		}
	//	}
	//	if (error)
	//		exit(0);

	//	delete[] tmptetlist;
	//	delete[] tmpidxlist;
	//}

	//{
	//	bool error = false;
	//	tethandle* tmptetlist = new tethandle[t_recordoldtetlist.size()];
	//	thrust::copy(t_recordoldtetlist.begin(), t_recordoldtetlist.end(), tmptetlist);
	//	int* tmpidxlist = new int[t_recordoldtetidx.size()];
	//	thrust::copy(t_recordoldtetidx.begin(), t_recordoldtetidx.end(), tmpidxlist);

	//	printf("after initial cavity recordoldtetlist: list size = %d, idx size = %d\n",
	//		t_recordoldtetlist.size(), t_recordoldtetidx.size());
	//	for (int i = 0; i < t_recordoldtetlist.size(); i++)
	//	{
	//		tethandle tmp = tmptetlist[i];
	//		int tmpidx = tmpidxlist[i];
	//		//printf("%d, %d, %d\n", tmp.id, tmp.ver, tmpidx);
	//		if (tmp.id < 0 || tmpidx < 0)
	//		{
	//			printf("tet.id = %d, threadId = %d\n", tmp.id, tmpidx);
	//			error = true;
	//			break;
	//		}
	//	}
	//	if (error)
	//		exit(0);

	//	delete[] tmptetlist;
	//	delete[] tmpidxlist;
	//}
#endif

	// Expand Initial Cavity
	// Every iteration, test if current tet in cavetetlist is included in cavity
	// If it is, expand cavetetlist and caveoldtetlist, otherwise expand cavebdrylist
	int cavetetcurstartindex = 0;
	int cavetetstartindex = t_cavetetlist.size();
	int caveoldtetstartindex = t_caveoldtetlist.size();
	int cavebdrystartindex = t_cavebdrylist.size();
	int cavetetexpandsize = cavetetstartindex, caveoldtetexpandsize, cavebdryexpandsize;

	numberofthreads = cavetetexpandsize; // each thread works on one tet in cavetetlist
	numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);

	IntD t_cavetetexpandsize(numberofthreads, 0);
	IntD t_caveoldtetexpandsize(numberofthreads, 0);
	IntD t_cavebdryexpandsize(numberofthreads, 0);
	IntD t_cavetetexpandindices(numberofthreads, -1);
	IntD t_caveoldtetexpandindices(numberofthreads, -1);
	IntD t_cavebdryexpandindices(numberofthreads, -1);

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        cavity expanding vector initialization = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["cavity expanding vector initialization"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

#ifdef GQM3D_LOOP_PROFILING
	double cavity_total_time = 0;
#endif
	int iteration = 0;
	while (true)
	{
		if (behavior->cavitymode == 1 && iteration > behavior->maxcavity) // Too large cavities. Stop and mark as unsplittable elements
		{
			kernelLargeCavityCheck << < numberofblocks, BLOCK_SIZE >> >(
				thrust::raw_pointer_cast(&t_insertidxlist[0]),
				thrust::raw_pointer_cast(&t_insertptlist[0]),
				thrust::raw_pointer_cast(&t_cavetetidx[0]),
				thrust::raw_pointer_cast(&t_tristatus[0]),
				thrust::raw_pointer_cast(&t_tetstatus[0]),
				thrust::raw_pointer_cast(&t_threadmarker[0]),
				cavetetcurstartindex,
				numberofthreads
				);

#ifdef GQM3D_DEBUG
			gpuErrchk(hipPeekAtLastError());
			gpuErrchk(hipDeviceSynchronize());
#endif

			break;
		}
		else if (behavior->cavitymode == 2 && iteration > behavior->mincavity)
		{
			int oldnumofthreads = numberofthreads;

			kernelMarkCavityReuse << < numberofblocks, BLOCK_SIZE >> >(
				thrust::raw_pointer_cast(&t_insertidxlist[0]),
				thrust::raw_pointer_cast(&t_cavetetidx[0]),
				thrust::raw_pointer_cast(&t_tristatus[0]),
				thrust::raw_pointer_cast(&t_tetstatus[0]),
				thrust::raw_pointer_cast(&t_threadmarker[0]),
				cavetetcurstartindex,
				numberofthreads
				);

#ifdef GQM3D_DEBUG
			gpuErrchk(hipPeekAtLastError());
			gpuErrchk(hipDeviceSynchronize());
#endif

			numberofthreads = t_caveoldtetlist.size();
			numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);
			kernelMarkOldtetlist << < numberofblocks, BLOCK_SIZE >> >(
				thrust::raw_pointer_cast(&t_caveoldtetlist[0]),
				thrust::raw_pointer_cast(&t_caveoldtetidx[0]),
				thrust::raw_pointer_cast(&t_insertidxlist[0]),
				thrust::raw_pointer_cast(&t_tristatus[0]),
				thrust::raw_pointer_cast(&t_tetstatus[0]),
				thrust::raw_pointer_cast(&t_threadmarker[0]),
				numberofthreads
				);

#ifdef GQM3D_DEBUG
			gpuErrchk(hipPeekAtLastError());
			gpuErrchk(hipDeviceSynchronize());
#endif

			int expandrecordsize = thrust::count_if(t_caveoldtetlist.begin(), t_caveoldtetlist.end(), isInvalidTetHandle());
			//printf("expandrecordsize = %d\n", expandrecordsize);
			int oldrecordsize = t_recordoldtetidx.size();
			t_recordoldtetlist.resize(oldrecordsize + expandrecordsize);
			t_recordoldtetidx.resize(oldrecordsize + expandrecordsize);
			auto first_old_iter = thrust::make_zip_iterator(thrust::make_tuple(t_caveoldtetlist.begin(), t_caveoldtetidx.begin()));
			auto last_old_iter = thrust::make_zip_iterator(thrust::make_tuple(t_caveoldtetlist.end(), t_caveoldtetidx.end()));
			auto first_record_iter = 
				thrust::make_zip_iterator(
					thrust::make_tuple(
						t_recordoldtetlist.begin() + oldrecordsize, 
						t_recordoldtetidx.begin() + oldrecordsize));
			auto last_record_iter = 
				thrust::copy_if(first_old_iter, last_old_iter, first_record_iter, isCavityTupleToRecord());
			//printf("distance = %d\n", thrust::distance(first_record_iter, last_record_iter));

			numberofthreads = expandrecordsize;
			numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);
			kernelSetRecordOldtet << < numberofblocks, BLOCK_SIZE >> >(
				thrust::raw_pointer_cast(&t_recordoldtetlist[0]),
				thrust::raw_pointer_cast(&t_recordoldtetidx[0]),
				thrust::raw_pointer_cast(&t_insertidxlist[0]),
				oldrecordsize,
				numberofthreads
				);

#ifdef GQM3D_DEBUG
			//{
			//	bool error = false;
			//	tethandle* tmptetlist = new tethandle[t_recordoldtetlist.size()];
			//	thrust::copy(t_recordoldtetlist.begin(), t_recordoldtetlist.end(), tmptetlist);
			//	int* tmpidxlist = new int[t_recordoldtetidx.size()];
			//	thrust::copy(t_recordoldtetidx.begin(), t_recordoldtetidx.end(), tmpidxlist);

			//	printf("after cavity expanding recordoldtetlist: list size = %d, idx size = %d\n",
			//		t_recordoldtetlist.size(), t_recordoldtetidx.size());
			//	for (int i = 0; i < t_recordoldtetlist.size(); i++)
			//	{
			//		tethandle tmp = tmptetlist[i];
			//		int tmpidx = tmpidxlist[i];
			//		//printf("%d, %d, %d\n", tmp.id, tmp.ver, tmpidx);
			//		if (tmp.id < 0 || tmpidx < 0)
			//		{
			//			printf("tet.id = %d, threadId = %d\n", tmp.id, tmpidx);
			//			error = true;
			//			break;
			//		}
			//	}
			//	if (error)
			//		exit(0);

			//	delete[] tmptetlist;
			//	delete[] tmpidxlist;
			//}
#endif

			numberofthreads = oldnumofthreads;
			numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);
			kernelMarkLargeCavityAsLoser << < numberofblocks, BLOCK_SIZE >> >(
				thrust::raw_pointer_cast(&t_cavetetidx[0]),
				thrust::raw_pointer_cast(&t_threadmarker[0]),
				cavetetcurstartindex,
				numberofthreads
				);

#ifdef GQM3D_DEBUG
			gpuErrchk(hipPeekAtLastError());
			gpuErrchk(hipDeviceSynchronize());
#endif

			//exit(0);

			break;
		}

#ifdef GQM3D_LOOP_PROFILING
		double iter_total_time = 0;
#ifdef GQM3D_LOOP_PROFILING_VERBOSE
		printf("          iteration = %d, survivors = %d, number of tuples to check = %d\n",
			iteration,
			thrust::count_if(t_threadmarker.begin(), t_threadmarker.end(), isNotNegativeInt()),
			numberofthreads);
#endif
		hipDeviceSynchronize();
		ltv[0] = clock();
#endif

		// Check if current tet is included in cavity
		kernelCavityExpandingCheck << < numberofblocks, BLOCK_SIZE >> >(
			thrust::raw_pointer_cast(&t_cavetetidx[0]),
			thrust::raw_pointer_cast(&t_pointlist[0]),
			thrust::raw_pointer_cast(&t_weightlist[0]),
			thrust::raw_pointer_cast(&t_tetlist[0]),
			thrust::raw_pointer_cast(&t_neighborlist[0]),
			thrust::raw_pointer_cast(&t_insertptlist[0]),
			thrust::raw_pointer_cast(&t_cavetetlist[0]),
			thrust::raw_pointer_cast(&t_cavetetexpandsize[0]),
			thrust::raw_pointer_cast(&t_caveoldtetexpandsize[0]),
			thrust::raw_pointer_cast(&t_cavebdryexpandsize[0]),
			thrust::raw_pointer_cast(&t_threadmarker[0]),
			thrust::raw_pointer_cast(&t_priority[0]),
			thrust::raw_pointer_cast(&t_tetmarker[0]),
			cavetetcurstartindex,
			numberofthreads
			);

#ifdef GQM3D_DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_LOOP_PROFILING
		hipDeviceSynchronize();
		ltv[1] = clock();
		looptimer["kernelCavityExpandingCheck"] += (REAL)(ltv[1] - ltv[0]);
		iter_total_time += (REAL)(ltv[1] - ltv[0]);
#ifdef GQM3D_LOOP_PROFILING_VERBOSE
		printf("            kernelCavityExpandingCheck time = %lf\n", (REAL)(ltv[1] - ltv[0]));
#endif
		ltv[0] = ltv[1];
#endif

		kernelCorrectExpandingSize << < numberofblocks, BLOCK_SIZE >> > (
			thrust::raw_pointer_cast(&t_cavetetidx[0]),
			thrust::raw_pointer_cast(&t_cavetetexpandsize[0]),
			thrust::raw_pointer_cast(&t_caveoldtetexpandsize[0]),
			thrust::raw_pointer_cast(&t_cavebdryexpandsize[0]),
			thrust::raw_pointer_cast(&t_threadmarker[0]),
			cavetetcurstartindex,
			numberofthreads
			);

#ifdef GQM3D_DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_LOOP_PROFILING
		hipDeviceSynchronize();
		ltv[1] = clock();
		looptimer["kernelCorrectExpandingSize"] += (REAL)(ltv[1] - ltv[0]);
		iter_total_time += (REAL)(ltv[1] - ltv[0]);
#ifdef GQM3D_LOOP_PROFILING_VERBOSE
		printf("            kernelCorrectExpandingSize time = %lf\n", (REAL)(ltv[1] - ltv[0]));
#endif
		ltv[0] = ltv[1];
#endif

		thrust::exclusive_scan(
			thrust::make_zip_iterator(thrust::make_tuple(t_cavetetexpandsize.begin(), t_caveoldtetexpandsize.begin(), t_cavebdryexpandsize.begin())),
			thrust::make_zip_iterator(thrust::make_tuple(t_cavetetexpandsize.end(), t_caveoldtetexpandsize.end(), t_cavebdryexpandsize.end())),
			thrust::make_zip_iterator(thrust::make_tuple(t_cavetetexpandindices.begin(), t_caveoldtetexpandindices.begin(), t_cavebdryexpandindices.begin())),
			thrust::make_tuple(0, 0, 0),
			PrefixSumTupleOP());

#ifdef GQM3D_LOOP_PROFILING
		hipDeviceSynchronize();
		ltv[1] = clock();
		looptimer["exclusive_scan for expanding"] += (REAL)(ltv[1] - ltv[0]);
		iter_total_time += (REAL)(ltv[1] - ltv[0]);
#ifdef GQM3D_LOOP_PROFILING_VERBOSE
		printf("            exclusive_scan for expanding time = %lf\n", (REAL)(ltv[1] - ltv[0]));
#endif
		ltv[0] = ltv[1];
#endif

		// Count expanding sizes
		cavetetexpandsize = t_cavetetexpandindices[numberofthreads - 1] + t_cavetetexpandsize[numberofthreads - 1];
		caveoldtetexpandsize = t_caveoldtetexpandindices[numberofthreads - 1] + t_caveoldtetexpandsize[numberofthreads - 1];
		cavebdryexpandsize = t_cavebdryexpandindices[numberofthreads - 1] + t_cavebdryexpandsize[numberofthreads - 1];

#ifdef GQM3D_LOOP_PROFILING
		hipDeviceSynchronize();
		ltv[1] = clock();
		looptimer["count sizes for expanding"] += (REAL)(ltv[1] - ltv[0]);
#ifdef GQM3D_LOOP_PROFILING_VERBOSE
		printf("            count sizes for expanding time = %lf\n", (REAL)(ltv[1] - ltv[0]));
#endif
		ltv[0] = ltv[1];
#endif

#ifdef GQM3D_DEBUG
		//if (behavior->verbose >= 2)
		//	printf("          Iteration = %d, expand tet = %d, oldtet = %d, bdrytet = %d, survivor = %d\n",
		//		iteration, cavetetexpandsize, caveoldtetexpandsize, cavebdryexpandsize,
		//		thrust::count_if(t_threadmarker.begin(), t_threadmarker.end(), isNotNegativeInt()));
#endif

		// Prepare memeory
		oldsize = t_cavetetlist.size();
		newsize = oldsize + cavetetexpandsize;
		t_cavetetlist.resize(newsize);
		t_cavetetidx.resize(newsize);
		oldsize = t_caveoldtetlist.size();
		newsize = oldsize + caveoldtetexpandsize;
		t_caveoldtetlist.resize(newsize);
		t_caveoldtetidx.resize(newsize);
		oldsize = t_cavebdrylist.size();
		newsize = oldsize + cavebdryexpandsize;
		t_cavebdrylist.resize(newsize);
		t_cavebdryidx.resize(newsize);

#ifdef GQM3D_LOOP_PROFILING
		hipDeviceSynchronize();
		ltv[1] = clock();
		looptimer["resize vector for expanding"] += (REAL)(ltv[1] - ltv[0]);
		iter_total_time += (REAL)(ltv[1] - ltv[0]);
#ifdef GQM3D_LOOP_PROFILING_VERBOSE
		printf("            resize vector for expanding time = %lf\n", (REAL)(ltv[1] - ltv[0]));
#endif
		ltv[0] = ltv[1];
#endif

		kernelCavityExpandingMarkAndAppend << < numberofblocks, BLOCK_SIZE >> >(
			thrust::raw_pointer_cast(&t_cavetetidx[0]),
			thrust::raw_pointer_cast(&t_neighborlist[0]),
			thrust::raw_pointer_cast(&t_cavetetlist[0]),
			thrust::raw_pointer_cast(&t_cavetetexpandsize[0]),
			thrust::raw_pointer_cast(&t_cavetetexpandindices[0]),
			cavetetstartindex,
			thrust::raw_pointer_cast(&t_caveoldtetlist[0]),
			thrust::raw_pointer_cast(&t_caveoldtetidx[0]),
			thrust::raw_pointer_cast(&t_caveoldtetexpandsize[0]),
			thrust::raw_pointer_cast(&t_caveoldtetexpandindices[0]),
			caveoldtetstartindex,
			thrust::raw_pointer_cast(&t_cavebdrylist[0]),
			thrust::raw_pointer_cast(&t_cavebdryidx[0]),
			thrust::raw_pointer_cast(&t_cavebdryexpandsize[0]),
			thrust::raw_pointer_cast(&t_cavebdryexpandindices[0]),
			cavebdrystartindex,
			thrust::raw_pointer_cast(&t_threadmarker[0]),
			cavetetcurstartindex,
			numberofthreads
			);

#ifdef GQM3D_DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_LOOP_PROFILING
		hipDeviceSynchronize();
		ltv[1] = clock();
		looptimer["kernelCavityExpandingMarkAndAppend"] += (REAL)(ltv[1] - ltv[0]);
		iter_total_time += (REAL)(ltv[1] - ltv[0]);
#ifdef GQM3D_LOOP_PROFILING_VERBOSE
		printf("            kernelCavityExpandingMarkAndAppend time = %lf\n", (REAL)(ltv[1] - ltv[0]));
#endif
		ltv[0] = ltv[1];
#endif

		// Update working thread list
		numberofthreads = cavetetexpandsize;
		iteration++;
		if (numberofthreads == 0)
			break;

		// Update variables
		numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);
		cavetetcurstartindex = cavetetstartindex;
		cavetetstartindex = t_cavetetlist.size();
		caveoldtetstartindex = t_caveoldtetlist.size();
		cavebdrystartindex = t_cavebdrylist.size();

		// Reset expanding lists
		t_cavetetexpandsize.resize(numberofthreads);
		thrust::fill(t_cavetetexpandsize.begin(), t_cavetetexpandsize.end(), 0);
		t_cavetetexpandindices.resize(numberofthreads);

		t_caveoldtetexpandsize.resize(numberofthreads);
		thrust::fill(t_caveoldtetexpandsize.begin(), t_caveoldtetexpandsize.end(), 0);
		t_caveoldtetexpandindices.resize(numberofthreads);

		t_cavebdryexpandsize.resize(numberofthreads);
		thrust::fill(t_cavebdryexpandsize.begin(), t_cavebdryexpandsize.end(), 0);
		t_cavebdryexpandindices.resize(numberofthreads);

#ifdef GQM3D_LOOP_PROFILING
		hipDeviceSynchronize();
		ltv[1] = clock();
		looptimer["prepare memory for next expanding"] += (REAL)(ltv[1] - ltv[0]);
		iter_total_time += (REAL)(ltv[1] - ltv[0]);
		cavity_total_time += iter_total_time;
#ifdef GQM3D_LOOP_PROFILING_VERBOSE
		printf("            prepare memory for next expanding time = %lf\n", (REAL)(ltv[1] - ltv[0]));
		printf("          total time = %lf\n", iter_total_time);
#endif
		ltv[0] = ltv[1];
#endif
	}
#ifdef GQM3D_LOOP_PROFILING
#ifdef GQM3D_LOOP_PROFILING_VERBOSE
	printf("          cavity growing total time = %lf\n", cavity_total_time);
#endif
#endif

#ifdef GQM3D_DEBUG
	//{
	//	bool error = false;
	//	tethandle* tmptetlist = new tethandle[t_caveoldtetlist.size()];
	//	thrust::copy(t_caveoldtetlist.begin(), t_caveoldtetlist.end(), tmptetlist);
	//	int* tmpidxlist = new int[t_caveoldtetidx.size()];
	//	thrust::copy(t_caveoldtetidx.begin(), t_caveoldtetidx.end(), tmpidxlist);

	//	printf("after cavity expanding caveoldtetlist:\n");
	//	for (int i = 0; i < t_caveoldtetlist.size(); i++)
	//	{
	//		tethandle tmp = tmptetlist[i];
	//		int tmpidx = tmpidxlist[i];
	//		//printf("%d, %d, %d\n", tmp.id, tmp.ver, tmpidx);
	//		if (tmp.id < 0)
	//		{
	//			printf("tet.id = %d, threadId = %d\n", tmp.id, tmpidx);
	//			error = true;
	//			break;
	//		}
	//	}
	//	//if (error)
	//	//	exit(0);

	//	delete[] tmptetlist;
	//	delete[] tmpidxlist;
	//}
#endif

#ifdef GQM3D_CHECKMEMORY
	printf("Before release cavity memory\n");
	hipDeviceSynchronize();
	gpuMemoryCheck();
#endif

	// Update cavetet, caveoldtet, cavebdry sizes and factors
	behavior->cavetetsizefac = t_cavetetlist.size() * 1.0 / behavior->cavetetsize + 0.02;
	behavior->cavetetsize = t_cavetetlist.size();

	behavior->caveoldtetsizefac = t_caveoldtetlist.size() * 1.0 / behavior->caveoldtetsize + 0.02;
	behavior->caveoldtetsize = t_caveoldtetlist.size();

	behavior->cavebdrysizefac = t_cavebdrylist.size() * 1.0 / behavior->cavebdrysize + 0.02;
	behavior->cavebdrysize = t_cavebdrylist.size();

	if (behavior->filterstatus == 3)
	{
		behavior->cavetetsizefac = 1.1;
		behavior->caveoldtetsizefac = 1.1;
		behavior->cavebdrysizefac = 1.1;
		behavior->filterstatus = 1;
	}

	freeVec(t_cavetetlist);
	freeVec(t_cavetetidx);
	freeVec(t_cavetetexpandsize);
	freeVec(t_caveoldtetexpandsize);
	freeVec(t_cavebdryexpandsize);
	freeVec(t_cavetetexpandindices);
	freeVec(t_caveoldtetexpandindices);
	freeVec(t_cavebdryexpandindices);

#ifdef GQM3D_DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        cavity expanding time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["cavity expanding"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	// Update working threadlist to winners
	numberofthreads = updateActiveListByMarker(t_threadmarker, t_threadlist, t_threadmarker.size());
	//numberofthreads = thrust::count_if(t_threadmarker.begin(), t_threadmarker.end(), isNotNegativeInt());
	numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);
	if (behavior->verbose >= 1)
		printf("        After expanding cavity, numberofthreads = %d(#%d, #%d, #%d), total expanding iteration = %d\n",
			numberofthreads,
			thrust::count(t_threadmarker.begin(), t_threadmarker.end(), 0),
			thrust::count(t_threadmarker.begin(), t_threadmarker.end(), 1),
			thrust::count(t_threadmarker.begin(), t_threadmarker.end(), 2),
			iteration);
	if (numberofthreads == 0)
		return 1;

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        update working threadlist time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["update working thread list"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	// Remove elements whose owners lost from cavity lists
#ifdef GQM3D_DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
	printf("Before removing losers: caveoldtet size = %d, cavebdry size = %d\n",
		t_caveoldtetlist.size(), t_cavebdrylist.size());
#endif

	numberofthreads = t_caveoldtetlist.size();
	numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);

	// Mark adjacent cavities as losers and collect subfaces
	TriHandleD t_cavetetshlist;
	IntD t_cavetetshidx;
	IntD t_cavetetshsize(numberofthreads, 0), t_cavetetshindices(numberofthreads, -1);
	int cavetetshsize;

#ifdef GQM3D_DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        subface vector initialization = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["subface vector initialization"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	kernelMarkAdjacentCavitiesAndCountSubfaces << < numberofblocks, BLOCK_SIZE >> >(
		thrust::raw_pointer_cast(&t_caveoldtetlist[0]),
		thrust::raw_pointer_cast(&t_caveoldtetidx[0]),
		thrust::raw_pointer_cast(&t_neighborlist[0]),
		thrust::raw_pointer_cast(&t_tet2trilist[0]),
		thrust::raw_pointer_cast(&t_cavetetshsize[0]),
		thrust::raw_pointer_cast(&t_priority[0]),
		thrust::raw_pointer_cast(&t_tetmarker[0]),
		thrust::raw_pointer_cast(&t_threadmarker[0]),
		numberofthreads
		);

#ifdef GQM3D_DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        kernelMarkAdjacentCavitiesAndCountSubfaces = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["kernelMarkAdjacentCavitiesAndCountSubfaces"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	kernelCorrectSubfaceSizes << < numberofblocks, BLOCK_SIZE >> > (
		thrust::raw_pointer_cast(&t_caveoldtetidx[0]),
		thrust::raw_pointer_cast(&t_cavetetshsize[0]),
		thrust::raw_pointer_cast(&t_threadmarker[0]),
		numberofthreads
		);

#ifdef GQM3D_DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        kernelCorrectSubfaceSizes = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["kernelCorrectSubfaceSizes"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	thrust::exclusive_scan(t_cavetetshsize.begin(), t_cavetetshsize.end(), t_cavetetshindices.begin());
	cavetetshsize = t_cavetetshindices[numberofthreads - 1] + t_cavetetshsize[numberofthreads - 1];
	t_cavetetshlist.resize(cavetetshsize);
	t_cavetetshidx.resize(cavetetshsize);

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        prepare subface vector time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["prepare subface vector"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	kernelAppendCavitySubfaces << < numberofblocks, BLOCK_SIZE >> >(
		thrust::raw_pointer_cast(&t_caveoldtetlist[0]),
		thrust::raw_pointer_cast(&t_caveoldtetidx[0]),
		thrust::raw_pointer_cast(&t_neighborlist[0]),
		thrust::raw_pointer_cast(&t_tet2trilist[0]),
		thrust::raw_pointer_cast(&t_cavetetshlist[0]),
		thrust::raw_pointer_cast(&t_cavetetshidx[0]),
		thrust::raw_pointer_cast(&t_cavetetshsize[0]),
		thrust::raw_pointer_cast(&t_cavetetshindices[0]),
		thrust::raw_pointer_cast(&t_tetmarker[0]),
		numberofthreads
		);

#ifdef GQM3D_DEBUG
	printf("cavetetshsize = %d\n", cavetetshsize);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
	//{
	//	for (int i = 0; i < t_cavetetshlist.size(); i++)
	//	{
	//		trihandle tmp = t_cavetetshlist[i];
	//		int tmpidx = t_cavetetshidx[i];
	//		printf("%d - %d\n", tmp.id, tmpidx);
	//	}
	//}
#endif

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        kernelAppendCavitySubfaces time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["kernelAppendCavitySubfaces"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	numberofthreads = cavetetshsize;
	if (numberofthreads > 0)
	{
		numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);

		t_cavetetshsize.resize(cavetetshsize); // indicate encroached: 0: no 1: yes
		thrust::fill(t_cavetetshsize.begin(), t_cavetetshsize.end(), 0);

		kernelCheckSubfaceEncroachment_Phase1 << < numberofblocks, BLOCK_SIZE >> > (
			thrust::raw_pointer_cast(&t_cavetetshlist[0]),
			thrust::raw_pointer_cast(&t_cavetetshidx[0]),
			thrust::raw_pointer_cast(&t_insertidxlist[0]),
			thrust::raw_pointer_cast(&t_insertptlist[0]),
			thrust::raw_pointer_cast(&t_pointlist[0]),
			thrust::raw_pointer_cast(&t_weightlist[0]),
			thrust::raw_pointer_cast(&t_trifacelist[0]),
			thrust::raw_pointer_cast(&t_trifacecent[0]),
			thrust::raw_pointer_cast(&t_tri2tetlist[0]),
			thrust::raw_pointer_cast(&t_tristatus[0]),
			thrust::raw_pointer_cast(&t_tetstatus[0]),
			thrust::raw_pointer_cast(&t_cavetetshsize[0]),
			thrust::raw_pointer_cast(&t_tetmarker[0]),
			thrust::raw_pointer_cast(&t_threadmarker[0]),
			numberofthreads
			);

#ifdef GQM3D_DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
		//{
		//	for (int i = 0; i < t_cavetetshsize.size(); i++)
		//	{
		//		int marker = t_cavetetshsize[i];
		//		printf("%d ", marker);
		//	}
		//	printf("\n");
		//}
#endif

		kernelCheckSubfaceEncroachment_Phase2 << < numberofblocks, BLOCK_SIZE >> > (
			thrust::raw_pointer_cast(&t_cavetetshidx[0]),
			thrust::raw_pointer_cast(&t_insertidxlist[0]),
			thrust::raw_pointer_cast(&t_tetstatus[0]),
			thrust::raw_pointer_cast(&t_cavetetshsize[0]),
			thrust::raw_pointer_cast(&t_threadmarker[0]),
			numberofthreads
			);

#ifdef GQM3D_DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif
	}

	// Refinement elements check
	// The new point is inserted by Delaunay refinement, i.e., it is the 
	//   circumcenter of a tetrahedron, or a subface, or a segment.
	//   Do not insert this point if the tetrahedron, or subface, or segment
	//   is not inside the final cavity.
	numberofthreads = t_threadlist.size();
	numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);
	kernelValidateRefinementElements << < numberofblocks, BLOCK_SIZE >> >(
		thrust::raw_pointer_cast(&t_insertidxlist[0]),
		thrust::raw_pointer_cast(&t_searchtet[0]),
		thrust::raw_pointer_cast(&t_neighborlist[0]),
		thrust::raw_pointer_cast(&t_tri2tetlist[0]),
		thrust::raw_pointer_cast(&t_threadlist[0]),
		thrust::raw_pointer_cast(&t_tristatus[0]),
		thrust::raw_pointer_cast(&t_tetstatus[0]),
		thrust::raw_pointer_cast(&t_tetmarker[0]),
		thrust::raw_pointer_cast(&t_threadmarker[0]),
		numberofthreads
		);

#ifdef GQM3D_DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        kernelValidateRefinementElements time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["kernelValidateRefinementElements"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	if (insertmode == 1)
	{
		kernelRecomputeTrifaceCenter << <numberofblocks, BLOCK_SIZE >> > (
			thrust::raw_pointer_cast(&t_insertidxlist[0]),
			thrust::raw_pointer_cast(&t_aabbnodeleft[0]),
			thrust::raw_pointer_cast(&t_aabbnoderight[0]),
			thrust::raw_pointer_cast(&t_aabbnodebbs[0]),
			thrust::raw_pointer_cast(&t_aabbpmcoord[0]),
			thrust::raw_pointer_cast(&t_aabbpmbbs[0]),
			thrust::raw_pointer_cast(&t_trifacecent[0]),
			thrust::raw_pointer_cast(&t_tristatus[0]),
			thrust::raw_pointer_cast(&t_trifacepmt[0]),
			thrust::raw_pointer_cast(&t_tri2tetlist[0]),
			thrust::raw_pointer_cast(&t_pointlist[0]),
			thrust::raw_pointer_cast(&t_weightlist[0]),
			thrust::raw_pointer_cast(&t_tetlist[0]),
			thrust::raw_pointer_cast(&t_neighborlist[0]),
			thrust::raw_pointer_cast(&t_tetstatus[0]),
			thrust::raw_pointer_cast(&t_threadlist[0]),
			thrust::raw_pointer_cast(&t_threadmarker[0]),
			aabb_diglen,
			numoftriface,
			numberofthreads
			);

#ifdef GQM3D_DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_PROFILING
		hipDeviceSynchronize();
		tv[1] = clock();
		printf("        kernelRecomputeTrifaceCenter time = %f\n", (REAL)(tv[1] - tv[0]));
		inserttimer["kernelRecomputeTrifaceCenter"] += (REAL)(tv[1] - tv[0]);
		tv[0] = tv[1];
#endif
	}

	freeVec(t_priority);
	freeVec(t_cavetetshsize);
	freeVec(t_cavetetshindices);

	numberofwonfacets = thrust::count(t_threadmarker.begin(), t_threadmarker.end(), 1);
	numberofthreads = updateActiveListByMarker(t_threadmarker, t_threadlist, t_threadmarker.size());
	//numberofthreads = thrust::count_if(t_threadmarker.begin(), t_threadmarker.end(), isNotNegativeInt());
	if (behavior->verbose >= 1)
		printf("        After boundary, encroachment and validity checking, numberofthreads = %d(#%d, #%d, #%d)\n",
			numberofthreads,
			thrust::count(t_threadmarker.begin(), t_threadmarker.end(), 0),
			thrust::count(t_threadmarker.begin(), t_threadmarker.end(), 1),
			thrust::count(t_threadmarker.begin(), t_threadmarker.end(), 2));
	if (numberofthreads == 0)
	{
		// This should not happen
		//printf("Error: 0 threads after boundary checking!\n");
		//exit(0);
		return 1;
	}

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        update working threadlist time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["update working thread list"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

#ifdef GQM3D_CHECKMEMORY
	hipDeviceSynchronize();
	gpuMemoryCheck();
#endif

	if (behavior->cavitymode == 2)
	{
		// All winners complete their cavities, reset flag if needed
		kernelResetCavityReuse << < numberofblocks, BLOCK_SIZE >> > (
			thrust::raw_pointer_cast(&t_insertidxlist[0]),
			thrust::raw_pointer_cast(&t_threadlist[0]),
			thrust::raw_pointer_cast(&t_tristatus[0]),
			thrust::raw_pointer_cast(&t_tetstatus[0]),
			thrust::raw_pointer_cast(&t_threadmarker[0]),
			numberofthreads
			);

#ifdef GQM3D_DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif
	}

	// Remove losers from  caveoldtet, cavebdry and cavetetsh
	int newlistsize;

	numberofthreads = t_caveoldtetlist.size();
	numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);
	kernelSetCavityThreadIdx << < numberofblocks, BLOCK_SIZE >> > (
		thrust::raw_pointer_cast(&t_caveoldtetidx[0]),
		thrust::raw_pointer_cast(&t_threadmarker[0]),
		numberofthreads
		);

#ifdef GQM3D_DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

	typedef thrust::zip_iterator<thrust::tuple<TetHandleD::iterator, IntD::iterator>> ZipIterator;
	ZipIterator first_iterator = thrust::make_zip_iterator(thrust::make_tuple(t_caveoldtetlist.begin(), t_caveoldtetidx.begin()));
	auto last_iterator =
		thrust::remove_if(first_iterator,
			thrust::make_zip_iterator(thrust::make_tuple(t_caveoldtetlist.end(), t_caveoldtetidx.end())),
			isInvalidCavityTuple());
	newlistsize = thrust::distance(first_iterator, last_iterator);
	t_caveoldtetlist.resize(newlistsize);
	t_caveoldtetidx.resize(newlistsize);

#ifdef GQM3D_DEBUG
	//{
	//	bool error = false;
	//	tethandle* tmptetlist = new tethandle[t_caveoldtetlist.size()];
	//	thrust::copy(t_caveoldtetlist.begin(), t_caveoldtetlist.end(), tmptetlist);
	//	int* tmpidxlist = new int[t_caveoldtetidx.size()];
	//	thrust::copy(t_caveoldtetidx.begin(), t_caveoldtetidx.end(), tmpidxlist);

	//	printf("caveoldtetlist:\n");
	//	for (int i = 0; i < t_caveoldtetlist.size(); i++)
	//	{
	//		tethandle tmp = tmptetlist[i];
	//		int tmpidx = tmpidxlist[i];
	//		//printf("%d, %d, %d\n", tmp.id, tmp.ver, tmpidx);
	//		if (tmp.id < 0)
	//		{
	//			printf("tet.id = %d, threadId = %d\n", tmp.id, tmpidx);
	//			error = true;
	//		}
	//	}
	//	if (error)
	//		exit(0);

	//	delete[] tmptetlist;
	//	delete[] tmpidxlist;
	//}
#endif

	numberofthreads = t_cavebdrylist.size();
	numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);
	kernelSetCavityThreadIdx << < numberofblocks, BLOCK_SIZE >> > (
		thrust::raw_pointer_cast(&t_cavebdryidx[0]),
		thrust::raw_pointer_cast(&t_threadmarker[0]),
		numberofthreads
		);

#ifdef GQM3D_DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

	first_iterator = thrust::make_zip_iterator(thrust::make_tuple(t_cavebdrylist.begin(), t_cavebdryidx.begin()));
	last_iterator =
		thrust::remove_if(first_iterator,
			thrust::make_zip_iterator(thrust::make_tuple(t_cavebdrylist.end(), t_cavebdryidx.end())),
			isInvalidCavityTuple());
	newlistsize = thrust::distance(first_iterator, last_iterator);
	t_cavebdrylist.resize(newlistsize);
	t_cavebdryidx.resize(newlistsize);

	numberofthreads = t_cavetetshlist.size();
	if (numberofthreads > 0)
	{
		numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);
		kernelSetCavityThreadIdx << < numberofblocks, BLOCK_SIZE >> > (
			thrust::raw_pointer_cast(&t_cavetetshidx[0]),
			thrust::raw_pointer_cast(&t_threadmarker[0]),
			numberofthreads
			);
	}

#ifdef GQM3D_DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

	auto first_iterator_s = thrust::make_zip_iterator(thrust::make_tuple(t_cavetetshlist.begin(), t_cavetetshidx.begin()));
	auto last_iterator_s =
			thrust::remove_if(first_iterator_s,
				thrust::make_zip_iterator(thrust::make_tuple(t_cavetetshlist.end(), t_cavetetshidx.end())),
				isInvalidSubfaceTuple());
	newlistsize = thrust::distance(first_iterator_s, last_iterator_s);
	t_cavetetshlist.resize(newlistsize);
	t_cavetetshidx.resize(newlistsize);

#ifdef GQM3D_DEBUG
	printf("After removing losers: caveoldtet size = %d, cavebdry size = %d, cavetetsh size = %d\n", 
		t_caveoldtetlist.size(), t_cavebdrylist.size(), t_cavetetshlist.size());
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        remove cavity losers time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["remove cavity losers"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	// Remove duplicate boundary faces in t_cavebdrylist
	first_iterator = thrust::make_zip_iterator(thrust::make_tuple(t_cavebdrylist.begin(), t_cavebdryidx.begin()));
	last_iterator = thrust::make_zip_iterator(thrust::make_tuple(t_cavebdrylist.end(), t_cavebdryidx.end()));
	thrust::sort(first_iterator, last_iterator, CavityTupleComp());

	first_iterator = thrust::make_zip_iterator(thrust::make_tuple(t_cavebdrylist.begin(), t_cavebdryidx.begin()));
	last_iterator =
		thrust::unique(first_iterator, 
			thrust::make_zip_iterator(thrust::make_tuple(t_cavebdrylist.end(), t_cavebdryidx.end())),
			CavityTupleEqualTo());
	newlistsize = thrust::distance(first_iterator, last_iterator);
	t_cavebdrylist.resize(newlistsize);
	t_cavebdryidx.resize(newlistsize);

	/*printf("cavebdry list:\n");
	for (int i = 0; i < t_cavebdrylist.size(); i++)
	{
		tethandle tmp = t_cavebdrylist[i];
		int tmpidx = t_cavebdryidx[i];
		printf("%d, %d, %d\n", tmp.id, tmp.ver, tmpidx);
	}*/

	/*numberofthreads = t_cavebdrylist.size();
	numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);
	kernelSetDuplicateThreadIdx << < numberofblocks, BLOCK_SIZE >> > (
		thrust::raw_pointer_cast(&t_cavebdrylist[0]),
		thrust::raw_pointer_cast(&t_cavebdryidx[0]),
		numberofthreads
		);*/

#ifdef GQM3D_DEBUG
	//printf("After removing duplicate faces: cavebdry size = %d\n", thrust::count_if(t_cavebdryidx.begin(), t_cavebdryidx.end(), isNotNegativeInt()));
	printf("After removing duplicate faces: cavebdry size = %d\n", t_cavebdryidx.size());
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        remove duplicate bdry faces time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["remove duplicate bdry"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	// Insert points into list
	numberofthreads = t_threadlist.size();
	numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);
	
	oldsize = t_weightlist.size();
	int oldpointsize = oldsize;
	newsize = oldsize + numberofthreads;
	t_pointlist.resize(3 * newsize);
	t_weightlist.resize(newsize, 0.0); // 0 weight for new insert points
	t_pointtypelist.resize(newsize);
	t_pointpmt.resize(newsize, -1);

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        prepare new point vector time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["prepare new point vector"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	IntD t_threadpos(numofinsertpt, -1);
	kernelInsertNewPoints << < numberofblocks, BLOCK_SIZE >> >(
		thrust::raw_pointer_cast(&t_threadlist[0]),
		thrust::raw_pointer_cast(&t_pointlist[0]),
		thrust::raw_pointer_cast(&t_pointtypelist[0]),
		thrust::raw_pointer_cast(&t_pointpmt[0]),
		thrust::raw_pointer_cast(&t_trifacepmt[0]),
		thrust::raw_pointer_cast(&t_insertidxlist[0]),
		thrust::raw_pointer_cast(&t_insertptlist[0]),
		thrust::raw_pointer_cast(&t_threadmarker[0]),
		thrust::raw_pointer_cast(&t_threadpos[0]),
		oldpointsize,
		numberofthreads
		);

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        kernelInsertNewPoints time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["kernelInsertNewPoints"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

#ifdef GQM3D_DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

	// Create new tetrahedra to fill the cavity
	int tetexpandsize = t_cavebdrylist.size();
	numberofthreads = tetexpandsize;
	numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);

	IntD t_tetexpandsize;
	if (insertmode == 1 && numberofwonfacets > 0)
	{
		t_tetexpandsize.resize(tetexpandsize);
		thrust::fill(t_tetexpandsize.begin(), t_tetexpandsize.end(), 0);

		kernelCountNewTets << < numberofblocks, BLOCK_SIZE >> >(
			thrust::raw_pointer_cast(&t_cavebdryidx[0]),
			thrust::raw_pointer_cast(&t_tetexpandsize[0]),
			thrust::raw_pointer_cast(&t_threadmarker[0]),
			numberofthreads
			);

		numberofnewtets_facet = thrust::count(t_tetexpandsize.begin(), t_tetexpandsize.end(), 1);

#ifdef GQM3D_DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_PROFILING
		hipDeviceSynchronize();
		tv[1] = clock();
		printf("        kernelCountNewTets time = %f\n", (REAL)(tv[1] - tv[0]));
		inserttimer["kernelCountNewTets"] += (REAL)(tv[1] - tv[0]);
		tv[0] = tv[1];
#endif
	}

	freeVec(t_tetexpandsize);

#ifdef GQM3D_DEBUG
	printf("Tet expanding size = %d\n", tetexpandsize);
#endif

	IntD t_emptytetslots;
	int numberofemptyslot = updateEmptyTetList(t_tetstatus, t_emptytetslots);
	if (numberofemptyslot < tetexpandsize) // dont have enough empty slots, extend lists
	{
		oldsize = t_tetstatus.size();
		newsize = oldsize + tetexpandsize - numberofemptyslot;
		try
		{
			t_tetlist.resize(4 * newsize, -1);
			t_neighborlist.resize(4 * newsize, tethandle(-1, 11));
			t_tet2trilist.resize(4 * newsize, trihandle(-1, 0));
			t_tetstatus.resize(newsize, tetstatus(0));
		}
		catch (thrust::system_error &e)
		{
			// output an error message and exit
			std::cerr << "Error: " << e.what() << std::endl;
			exit(-1);
		}
		numberofemptyslot = updateEmptyTetList(t_tetstatus, t_emptytetslots);
	}

#ifdef GQM3D_DEBUG
	printf("numberofemptyslot = %d\n", numberofemptyslot);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        prepare new tet vector time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["prepare new tet vector"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	IntD t_newtetthreadindices(tetexpandsize, -1); // used to update tetstatus later on
	kernelInsertNewTets << < numberofblocks, BLOCK_SIZE >> > (
		thrust::raw_pointer_cast(&t_cavebdrylist[0]),
		thrust::raw_pointer_cast(&t_cavebdryidx[0]),
		thrust::raw_pointer_cast(&t_tristatus[0]),
		thrust::raw_pointer_cast(&t_tri2tetlist[0]),
		thrust::raw_pointer_cast(&t_tetlist[0]),
		thrust::raw_pointer_cast(&t_neighborlist[0]),
		thrust::raw_pointer_cast(&t_tet2trilist[0]),
		thrust::raw_pointer_cast(&t_tetstatus[0]),
		thrust::raw_pointer_cast(&t_emptytetslots[0]),
		thrust::raw_pointer_cast(&t_newtetthreadindices[0]),
		thrust::raw_pointer_cast(&t_threadmarker[0]),
		thrust::raw_pointer_cast(&t_threadpos[0]),
		oldpointsize,
		numberofthreads
		);

#ifdef GQM3D_DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        kernelInsertNewTets time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["kernelInsertNewTets"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	// Connect adjacent new tetrahedra together
	kernelConnectNewTetNeighbors << < numberofblocks, BLOCK_SIZE >> >(
		thrust::raw_pointer_cast(&t_cavebdrylist[0]),
		thrust::raw_pointer_cast(&t_cavebdryidx[0]),
		thrust::raw_pointer_cast(&t_tetlist[0]),
		thrust::raw_pointer_cast(&t_neighborlist[0]),
		thrust::raw_pointer_cast(&t_tetstatus[0]),
		thrust::raw_pointer_cast(&t_tetmarker[0]),
		numberofthreads
		);

#ifdef GQM3D_DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_CHECKMEMORY
	printf("After inserting new elements\n");
	hipDeviceSynchronize();
	gpuMemoryCheck();
#endif

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        kernelConnectNewTetNeighbors time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["kernelConnectNewTetNeighbors"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	// Update tristatus and tetstatus, and
	// check and add new trifaces
	int triexpandsize = 0;
	IntD t_triexpandsize(4 * tetexpandsize, 0);
	IntD t_triexpandindice(4 * tetexpandsize, 0);
	IntD t_tripmtidx(4 * tetexpandsize, -1);
	RealD t_trifaceipt(4 * 3 * tetexpandsize);
	IntD t_emptytrislots;

#ifdef GQM3D_CHECKMEMORY
	printf("After triface expanding vector initialization\n");
	hipDeviceSynchronize();
	gpuMemoryCheck();
	//checkVectorSize(t_triexpandsize, "t_triexpandsize", int);
	//checkVectorSize(t_triexpandindice, "t_triexpandindice", int);
	//checkVectorSize(t_trifaceipt, "t_trifaceipt", REAL);
	//checkVectorSize(t_emptytetslots, "t_emptytrislots", int);
#endif

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        tri and tet status vector initialization time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["tri and tet status vector initializatio"] += (REAL)(tv[1] - tv[0]);
	tv[0] = tv[1];
#endif

	if (behavior->aabbmode == 1)
	{
		//printf("tetexpandsize = %d\n", tetexpandsize);
		numberofthreads = tetexpandsize;
		numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);

		kernelUpdateTriAndTetStatus_Phase1 << <numberofblocks, BLOCK_SIZE >> > (
			thrust::raw_pointer_cast(&t_aabbnodeleft[0]),
			thrust::raw_pointer_cast(&t_aabbnoderight[0]),
			thrust::raw_pointer_cast(&t_aabbnodebbs[0]),
			thrust::raw_pointer_cast(&t_aabbpmcoord[0]),
			thrust::raw_pointer_cast(&t_aabbpmbbs[0]),
			thrust::raw_pointer_cast(&t_emptytetslots[0]),
			thrust::raw_pointer_cast(&t_newtetthreadindices[0]),
			thrust::raw_pointer_cast(&t_triexpandsize[0]),
			thrust::raw_pointer_cast(&t_trifaceipt[0]),
			thrust::raw_pointer_cast(&t_tripmtidx[0]),
			thrust::raw_pointer_cast(&t_pointlist[0]),
			thrust::raw_pointer_cast(&t_weightlist[0]),
			thrust::raw_pointer_cast(&t_pointpmt[0]),
			thrust::raw_pointer_cast(&t_tetlist[0]),
			thrust::raw_pointer_cast(&t_neighborlist[0]),
			thrust::raw_pointer_cast(&t_tetstatus[0]),
			thrust::raw_pointer_cast(&t_threadmarker[0]),
			criteria->cell_radius_edge_ratio,
			criteria->cell_size,
			aabb_diglen,
			aabb_pmnum,
			insertmode,
			behavior->aabbshortcut,
			numberofthreads
			);

#ifdef GQM3D_DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_PROFILING
		hipDeviceSynchronize();
		tv[1] = clock();
		printf("        kernelUpdateTriAndTetStatus_Phase1 time = %f\n", (REAL)(tv[1] - tv[0]));
		inserttimer["kernelUpdateTriAndTetStatus_Phase1"] += (REAL)(tv[1] - tv[0]);
		tv[0] = tv[1];
#endif

		//triexpandsize = thrust::reduce(t_triexpandsize.begin(), t_triexpandsize.end());
		thrust::exclusive_scan(t_triexpandsize.begin(), t_triexpandsize.end(), t_triexpandindice.begin());
		triexpandsize = t_triexpandindice[4 * tetexpandsize - 1] + t_triexpandsize[4 * tetexpandsize - 1];
#ifdef GQM3D_DEBUG
		printf("Tri expanding size = %d\n", triexpandsize);
#endif

		//IntD t_emptytrislots;
		numberofemptyslot = updateEmptyTriList(t_tristatus, t_emptytrislots);
		if (numberofemptyslot < triexpandsize) // dont have enough empty slots, extend lists
		{
			oldsize = t_tristatus.size();
			newsize = oldsize + triexpandsize - numberofemptyslot;
			try
			{
				t_trifacelist.resize(3 * newsize, -1);
				t_trifacecent.resize(3 * newsize);
				t_tri2tetlist.resize(2 * newsize, tethandle(-1, 11));
				t_tristatus.resize(newsize, tristatus(0));
				t_trifacepmt.resize(newsize, -1);
			}
			catch (thrust::system_error &e)
			{
				// output an error message and exit
				std::cerr << "Error: " << e.what() << std::endl;
				exit(-1);
			}
			numberofemptyslot = updateEmptyTriList(t_tristatus, t_emptytrislots);
		}

#ifdef GQM3D_DEBUG
		printf("numberofemptyslot = %d\n", numberofemptyslot);
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_PROFILING
		hipDeviceSynchronize();
		tv[1] = clock();
		printf("        prepare new subface vector time = %f\n", (REAL)(tv[1] - tv[0]));
		inserttimer["prepare new subface vector"] += (REAL)(tv[1] - tv[0]);
		tv[0] = tv[1];
#endif

		kernelUpdateTriAndTetStatus_Phase2 << <numberofblocks, BLOCK_SIZE >> > (
			thrust::raw_pointer_cast(&t_emptytetslots[0]),
			thrust::raw_pointer_cast(&t_triexpandsize[0]),
			thrust::raw_pointer_cast(&t_triexpandindice[0]),
			thrust::raw_pointer_cast(&t_emptytrislots[0]),
			thrust::raw_pointer_cast(&t_trifaceipt[0]),
			thrust::raw_pointer_cast(&t_tripmtidx[0]),
			thrust::raw_pointer_cast(&t_pointlist[0]),
			thrust::raw_pointer_cast(&t_pointtypelist[0]),
			thrust::raw_pointer_cast(&t_weightlist[0]),
			thrust::raw_pointer_cast(&t_trifacelist[0]),
			thrust::raw_pointer_cast(&t_trifacecent[0]),
			thrust::raw_pointer_cast(&t_tristatus[0]),
			thrust::raw_pointer_cast(&t_trifacepmt[0]),
			thrust::raw_pointer_cast(&t_tri2tetlist[0]),
			thrust::raw_pointer_cast(&t_tetlist[0]),
			thrust::raw_pointer_cast(&t_neighborlist[0]),
			thrust::raw_pointer_cast(&t_tet2trilist[0]),
			thrust::raw_pointer_cast(&t_tetstatus[0]),
			criteria->facet_angle,
			criteria->facet_size,
			criteria->facet_distance,
			numberofthreads
			);

#ifdef GQM3D_DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_PROFILING
		hipDeviceSynchronize();
		tv[1] = clock();
		printf("        kernelUpdateTriAndTetStatus_Phase2 time = %f\n", (REAL)(tv[1] - tv[0]));
		inserttimer["kernelUpdateTriAndTetStatus_Phase2"] += (REAL)(tv[1] - tv[0]);
		tv[0] = tv[1];
#endif
	}
	else if (behavior->aabbmode == 2)
	{
		int curhandlesize;
		TetHandleD::iterator last_iterator;
		IntD t_domaincount(tetexpandsize, 0);// counter for in/out domain test
		TetHandleD t_domainhandle;
		RealD t_domainsegment;
		IntD t_domainthreadlist;
		IntD t_domainnode;
		int winsize = tetexpandsize;

		int rechandlesize = behavior->aabbhandlesizefac * behavior->aabbhandlesize;
		double free_mb;
		getFreeMemory(free_mb);
		int maxreservesize = free_mb * 1024 * 1024 / 6 / sizeof(tethandle); // 1/6 of the available memory
																			//printf("rechandlesize = %d, maxreservesize = %d\n", rechandlesize, maxreservesize);
		if (rechandlesize < maxreservesize)
		{
			t_domainhandle.reserve(rechandlesize);
			t_domainnode.reserve(rechandlesize);
		}
		else
		{
			//printf("Enter memory save mode\n");
			t_domainhandle.reserve(maxreservesize);
			t_domainnode.reserve(maxreservesize);
			if (behavior->aabbwinsize == -1)
				behavior->aabbwinsize = tetexpandsize;
			else
				winsize = behavior->aabbwinsize; // set proper winsize
		}

		try
		{
			if (t_tetmarker.size() < 4 * tetexpandsize) // use temporarily for unique distance marking
				t_tetmarker.resize(4 * tetexpandsize);
			thrust::fill(t_tetmarker.begin(), t_tetmarker.begin() + 4 * tetexpandsize, MAXULL);
		}
		catch (thrust::system_error &e)
		{
			// output an error message and exit
			std::cerr << "Error: " << e.what() << std::endl;
			exit(-1);
		}

#ifdef GQM3D_PROFILING
		hipDeviceSynchronize();
		tv[1] = clock();
		printf("        domain vector initialization time = %f\n", (REAL)(tv[1] - tv[0]));
		inserttimer["domain vector initialization"] += (REAL)(tv[1] - tv[0]);
		tv[0] = clock();
#endif

		int offset = 0;
		int oldaabbhandlesize = behavior->aabbhandlesize;
		behavior->aabbhandlesize = 0;
		while (true)
		{
			numberofthreads = tetexpandsize - offset;
			if (numberofthreads > winsize)
				numberofthreads = winsize;
			else if (numberofthreads <= 0)
				break;

			//printf("tetexpandsize = %d, numberofthreads = %d, offset = %d, winsize = %d\n", tetexpandsize, numberofthreads, offset, winsize);

			numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);
			if (insertmode == 0)
			{
				t_domainhandle.resize(5 * numberofthreads);
			}
			else
			{
				if (numberofwonfacets == 0) // nothing to do
					break;
				t_domainhandle.resize(5 * numberofnewtets_facet);
			}
			thrust::fill(t_domainhandle.begin(), t_domainhandle.end(), tethandle(-1, 11));
			kernelInitDomainHandle << <numberofblocks, BLOCK_SIZE >> > (
				thrust::raw_pointer_cast(&t_emptytetslots[0]),
				thrust::raw_pointer_cast(&t_newtetthreadindices[0]),
				thrust::raw_pointer_cast(&t_domainhandle[0]),
				thrust::raw_pointer_cast(&t_domaincount[0]),
				thrust::raw_pointer_cast(&t_tetlist[0]),
				thrust::raw_pointer_cast(&t_neighborlist[0]),
				thrust::raw_pointer_cast(&t_tetstatus[0]),
				thrust::raw_pointer_cast(&t_threadmarker[0]),
				insertmode,
				offset,
				numberofthreads
				);

#ifdef GQM3D_DEBUG
			gpuErrchk(hipPeekAtLastError());
			gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_PROFILING
			hipDeviceSynchronize();
			tv[1] = clock();
			printf("        kernelInitDomainHandle time = %f\n", (REAL)(tv[1] - tv[0]));
			inserttimer["kernelInitDomainHandle"] += (REAL)(tv[1] - tv[0]);
			tv[0] = clock();
#endif

			last_iterator =
				thrust::remove_if(t_domainhandle.begin(), t_domainhandle.end(), isInvalidTetHandle());
			curhandlesize = thrust::distance(t_domainhandle.begin(), last_iterator);
			//printf("curhandlesize = %d\n", curhandlesize);
			if (curhandlesize == 0)
			{
				offset += winsize;
				continue;
			}

			try
			{
				t_domainsegment.resize(6 * curhandlesize);
				t_domainthreadlist.resize(curhandlesize); // thread indice list to store new tet thread indice
			}
			catch (thrust::system_error &e)
			{
				// output an error message and exit
				std::cerr << "Error: " << e.what() << std::endl;
				exit(-1);
			}

#ifdef GQM3D_PROFILING
			hipDeviceSynchronize();
			tv[1] = clock();
			printf("        prepare for domain segment vector time = %f\n", (REAL)(tv[1] - tv[0]));
			inserttimer["prepare for domain segment vector"] += (REAL)(tv[1] - tv[0]);
			tv[0] = clock();
#endif

			numberofthreads = curhandlesize;
			numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);
			kernelInitDomainSegment << <numberofblocks, BLOCK_SIZE >> > (
				thrust::raw_pointer_cast(&t_aabbnodebbs[0]),
				thrust::raw_pointer_cast(&t_aabbpmcoord[0]),
				thrust::raw_pointer_cast(&t_aabbpmbbs[0]),
				thrust::raw_pointer_cast(&t_domainhandle[0]),
				thrust::raw_pointer_cast(&t_domainsegment[0]),
				thrust::raw_pointer_cast(&t_domainthreadlist[0]),
				thrust::raw_pointer_cast(&t_triexpandsize[0]),
				thrust::raw_pointer_cast(&t_trifaceipt[0]),
				thrust::raw_pointer_cast(&t_tripmtidx[0]),
				thrust::raw_pointer_cast(&t_emptytetslots[0]),
				thrust::raw_pointer_cast(&t_pointlist[0]),
				thrust::raw_pointer_cast(&t_weightlist[0]),
				thrust::raw_pointer_cast(&t_pointpmt[0]),
				thrust::raw_pointer_cast(&t_tetlist[0]),
				thrust::raw_pointer_cast(&t_neighborlist[0]),
				aabb_diglen,
				aabb_pmnum,
				insertmode,
				behavior->aabbshortcut,
				numberofthreads
				);

#ifdef GQM3D_DEBUG
			gpuErrchk(hipPeekAtLastError());
			gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_PROFILING
			hipDeviceSynchronize();
			tv[1] = clock();
			printf("        kernelInitDomainSegment time = %f\n", (REAL)(tv[1] - tv[0]));
			inserttimer["kernelInitDomainSegment"] += (REAL)(tv[1] - tv[0]);
			tv[0] = clock();
#endif

			last_iterator = // remove degenerate cases and fast check cases
				thrust::remove_if(t_domainhandle.begin(), t_domainhandle.begin() + numberofthreads, isInvalidTetHandle());
			curhandlesize = thrust::distance(t_domainhandle.begin(), last_iterator);
			//printf("curhandlesize = %d\n", curhandlesize);
			if (curhandlesize == 0)
			{
				offset += winsize;
				continue;
			}
			t_domainnode.resize(t_domainhandle.size());
			thrust::fill(t_domainnode.begin(), t_domainnode.begin() + curhandlesize, 1);
			numberofthreads = curhandlesize;
			numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);

#ifdef GQM3D_PROFILING
			hipDeviceSynchronize();
			tv[1] = clock();
			printf("        prepare vector for domain search time = %f\n", (REAL)(tv[1] - tv[0]));
			inserttimer["prepare vector for domain search"] += (REAL)(tv[1] - tv[0]);
			tv[0] = clock();
#endif

#ifdef GQM3D_CHECKMEMORY
			printf("Domain search initialization\n");
			hipDeviceSynchronize();
			gpuMemoryCheck();
			//checkVectorSize(t_domaincount, "t_domaincount", int);
			//checkVectorSize(t_domainhandle, "t_domainhandle", tethandle);
			//checkVectorSize(t_domainnode, "t_domainnode", int);
			//checkVectorSize(t_domainsegment, "t_domainsegment", REAL);
			//checkVectorSize(t_domainthreadlist, "t_domainthreadlist", int);
#endif

			int numofemptyhandleslot;
			int domainexpanditer = 0;
			int maxhandlesize = curhandlesize;
			bool halfwinsize = false;
			while (true)
			{
				if (curhandlesize > maxhandlesize)
					maxhandlesize = curhandlesize;

#ifdef GQM3D_LOOP_PROFILING
				hipDeviceSynchronize();
				ltv[0] = clock();
#endif
				//printf("Domain search iteration = %d, curhandlesize = %d\n", domainexpanditer, curhandlesize);
				kernelDomainSegmentAndBoxCheck << <numberofblocks, BLOCK_SIZE >> > (
					thrust::raw_pointer_cast(&t_aabbnodebbs[0]),
					thrust::raw_pointer_cast(&t_domainhandle[0]),
					thrust::raw_pointer_cast(&t_domainnode[0]),
					thrust::raw_pointer_cast(&t_domainsegment[0]),
					numberofthreads
					);

#ifdef GQM3D_DEBUG
				gpuErrchk(hipPeekAtLastError());
				gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_LOOP_PROFILING
				hipDeviceSynchronize();
				ltv[1] = clock();
				looptimer["kernelDomainSegmentAndBoxCheck"] += (REAL)(ltv[1] - ltv[0]);
				ltv[0] = ltv[1];
#endif
				// remove the handles that do not intersect with node bounding boxes 
				typedef thrust::zip_iterator<thrust::tuple<TetHandleD::iterator, IntD::iterator>> ZipIterator;
				ZipIterator first_iterator = thrust::make_zip_iterator(thrust::make_tuple(t_domainhandle.begin(), t_domainnode.begin()));
				auto last_iterator =
					thrust::remove_if(first_iterator,
						thrust::make_zip_iterator(thrust::make_tuple(t_domainhandle.begin() + numberofthreads, t_domainnode.begin() + numberofthreads)),
						isInvalidDomainTuple());
				curhandlesize = thrust::distance(first_iterator, last_iterator);
				if (curhandlesize == 0)
					break;

#ifdef GQM3D_LOOP_PROFILING
				hipDeviceSynchronize();
				ltv[1] = clock();
				looptimer["remove_if for domain search"] += (REAL)(ltv[1] - ltv[0]);
				ltv[0] = ltv[1];
#endif

				// It seems that in some very rare case, t_domainnode would exit with positive elements
				if (domainexpanditer == aabb_level)
					break;

				// prepare enough space for new handles and nodes
				numofemptyhandleslot = t_domainhandle.size() - curhandlesize;
				if (numofemptyhandleslot < curhandlesize)
				{
					try
					{
						if (2 * curhandlesize > maxreservesize) // possible to run out of memory
						{
							// half the window size
							//printf("half the window size\n");
							winsize /= 2;
							halfwinsize = true;
							behavior->aabbwinsize = winsize;
							break;
						}

						t_domainhandle.resize(2 * curhandlesize);
						t_domainnode.resize(2 * curhandlesize);
					}
					catch (thrust::system_error &e)
					{
						// output an error message and exit
						std::cerr << "Error: " << e.what() << std::endl;
						exit(-1);
					}
				}

#ifdef GQM3D_LOOP_PROFILING
				hipDeviceSynchronize();
				ltv[1] = clock();
				looptimer["resize handle and node vector"] += (REAL)(ltv[1] - ltv[0]);
				ltv[0] = ltv[1];
#endif

				thrust::fill(t_domainhandle.begin() + curhandlesize, t_domainhandle.begin() + 2 * curhandlesize, tethandle(-1, 11));
				thrust::fill(t_domainnode.begin() + curhandlesize, t_domainnode.begin() + 2 * curhandlesize, 0);

#ifdef GQM3D_LOOP_PROFILING
				hipDeviceSynchronize();
				ltv[1] = clock();
				looptimer["fill handle and node vector"] += (REAL)(ltv[1] - ltv[0]);
				ltv[0] = ltv[1];
#endif

				numberofthreads = curhandlesize;
				numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);
				kernelDomainHandleAppend << <numberofblocks, BLOCK_SIZE >> > (
					thrust::raw_pointer_cast(&t_aabbnodeleft[0]),
					thrust::raw_pointer_cast(&t_aabbnoderight[0]),
					thrust::raw_pointer_cast(&t_domainhandle[0]),
					thrust::raw_pointer_cast(&t_domainnode[0]),
					numberofthreads
					);

#ifdef GQM3D_DEBUG
				gpuErrchk(hipPeekAtLastError());
				gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_LOOP_PROFILING
				hipDeviceSynchronize();
				ltv[1] = clock();
				looptimer["kernelDomainHandleAppend"] += (REAL)(ltv[1] - ltv[0]);
#endif

				curhandlesize = 2 * curhandlesize;
				numberofthreads = curhandlesize;
				numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);

				domainexpanditer++;
			}

			if (halfwinsize)
				continue;

#ifdef GQM3D_CHECKMEMORY
			printf("After Domain search\n");
			hipDeviceSynchronize();
			gpuMemoryCheck();
#endif

#ifdef GQM3D_PROFILING
			hipDeviceSynchronize();
			tv[1] = clock();
			printf("        domain search time = %f\n", (REAL)(tv[1] - tv[0]));
			inserttimer["domain search"] += (REAL)(tv[1] - tv[0]);
			tv[0] = clock();
#endif
			behavior->aabbhandlesize += maxhandlesize; // record the largest handle size to use in next iteration

			if (curhandlesize == 0)
			{
				offset += winsize;
				continue;
			}

			//printf("curhandlesize = %d, tetexpandsize = %d, t_tetmarker.size() = %d, t_insertptlist.size() = %d, oldhandlesize = %d, handlesizefac = %lf\n", 
			//	curhandlesize, tetexpandsize, t_tetmarker.size(), t_insertptlist.size(), behavior->aabbhandlesize, behavior->aabbhandlesizefac);

			numberofthreads = curhandlesize;
			numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);

#ifdef GQM3D_PROFILING
			hipDeviceSynchronize();
			tv[1] = clock();
			printf("        prepare vector for intersection marking time = %f\n", (REAL)(tv[1] - tv[0]));
			inserttimer["prepare vector for intersection marking"] += (REAL)(tv[1] - tv[0]);
			tv[0] = clock();
#endif

			kernelDomainSegmentAndPrimitiveCheck << <numberofblocks, BLOCK_SIZE >> > (
				thrust::raw_pointer_cast(&t_aabbpmcoord[0]),
				thrust::raw_pointer_cast(&t_aabbpmbbs[0]),
				thrust::raw_pointer_cast(&t_domainhandle[0]),
				thrust::raw_pointer_cast(&t_domainnode[0]),
				thrust::raw_pointer_cast(&t_domainsegment[0]),
				thrust::raw_pointer_cast(&t_domaincount[0]),
				thrust::raw_pointer_cast(&t_domainthreadlist[0]),
				thrust::raw_pointer_cast(&t_triexpandsize[0]),
				thrust::raw_pointer_cast(&t_emptytetslots[0]),
				thrust::raw_pointer_cast(&t_pointlist[0]),
				thrust::raw_pointer_cast(&t_weightlist[0]),
				thrust::raw_pointer_cast(&t_tetlist[0]),
				thrust::raw_pointer_cast(&t_tetmarker[0]),
				numberofthreads
				);

#ifdef GQM3D_DEBUG
			gpuErrchk(hipPeekAtLastError());
			gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_PROFILING
			hipDeviceSynchronize();
			tv[1] = clock();
			printf("        kernelDomainSegmentAndPrimitiveCheck time = %f\n", (REAL)(tv[1] - tv[0]));
			inserttimer["kernelDomainSegmentAndPrimitiveCheck"] += (REAL)(tv[1] - tv[0]);
			tv[0] = clock();
#endif

			kernelDomainSetTriCenter << <numberofblocks, BLOCK_SIZE >> > (
				thrust::raw_pointer_cast(&t_aabbpmcoord[0]),
				thrust::raw_pointer_cast(&t_aabbpmbbs[0]),
				thrust::raw_pointer_cast(&t_domainhandle[0]),
				thrust::raw_pointer_cast(&t_domainnode[0]),
				thrust::raw_pointer_cast(&t_domainsegment[0]),
				thrust::raw_pointer_cast(&t_domainthreadlist[0]),
				thrust::raw_pointer_cast(&t_tetmarker[0]),
				thrust::raw_pointer_cast(&t_trifaceipt[0]),
				thrust::raw_pointer_cast(&t_tripmtidx[0]),
				numberofthreads
				);

#ifdef GQM3D_DEBUG
			gpuErrchk(hipPeekAtLastError());
			gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_CHECKMEMORY
			printf("After intersection points calculation\n");
			hipDeviceSynchronize();
			gpuMemoryCheck();
#endif

#ifdef GQM3D_PROFILING
			hipDeviceSynchronize();
			tv[1] = clock();
			printf("        kernelDomainSetTriCenter time = %f\n", (REAL)(tv[1] - tv[0]));
			inserttimer["kernelDomainSetTriCenter"] += (REAL)(tv[1] - tv[0]);
			tv[0] = clock();
#endif

			offset += winsize;
		}

		if (oldaabbhandlesize != 0)
			behavior->aabbhandlesizefac = behavior->aabbhandlesize*1.0 / oldaabbhandlesize + 0.02;
		else
			behavior->aabbhandlesizefac = 1.002;

		if (behavior->filterstatus == 3)
		{
			behavior->aabbhandlesizefac = 1.5;
			behavior->filterstatus = 1;
		}

		thrust::exclusive_scan(t_triexpandsize.begin(), t_triexpandsize.end(), t_triexpandindice.begin());
		triexpandsize = t_triexpandindice[4 * tetexpandsize - 1] + t_triexpandsize[4 * tetexpandsize - 1];
#ifdef GQM3D_DEBUG
		printf("Tri expanding size = %d\n", triexpandsize);
#endif

		numberofemptyslot = updateEmptyTriList(t_tristatus, t_emptytrislots);
		if (numberofemptyslot < triexpandsize) // dont have enough empty slots, extend lists
		{
			oldsize = t_tristatus.size();
			newsize = oldsize + triexpandsize - numberofemptyslot;
			try
			{
				t_trifacelist.resize(3 * newsize, -1);
				t_trifacecent.resize(3 * newsize);
				t_tri2tetlist.resize(2 * newsize, tethandle(-1, 11));
				t_tristatus.resize(newsize, tristatus(0));
				t_trifacepmt.resize(newsize, -1);
			}
			catch (thrust::system_error &e)
			{
				// output an error message and exit
				std::cerr << "Error: " << e.what() << std::endl;
				exit(-1);
			}
			numberofemptyslot = updateEmptyTriList(t_tristatus, t_emptytrislots);
		}

#ifdef GQM3D_PROFILING
		hipDeviceSynchronize();
		tv[1] = clock();
		printf("        prepare new subface vector time = %f\n", (REAL)(tv[1] - tv[0]));
		inserttimer["prepare new subface vector"] += (REAL)(tv[1] - tv[0]);
		tv[0] = clock();
#endif

		numberofthreads = tetexpandsize;
		numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);
		kernelUpdateTriAndTetStatus_Phase2 << <numberofblocks, BLOCK_SIZE >> > (
			thrust::raw_pointer_cast(&t_emptytetslots[0]),
			thrust::raw_pointer_cast(&t_triexpandsize[0]),
			thrust::raw_pointer_cast(&t_triexpandindice[0]),
			thrust::raw_pointer_cast(&t_emptytrislots[0]),
			thrust::raw_pointer_cast(&t_trifaceipt[0]),
			thrust::raw_pointer_cast(&t_tripmtidx[0]),
			thrust::raw_pointer_cast(&t_pointlist[0]),
			thrust::raw_pointer_cast(&t_pointtypelist[0]),
			thrust::raw_pointer_cast(&t_weightlist[0]),
			thrust::raw_pointer_cast(&t_trifacelist[0]),
			thrust::raw_pointer_cast(&t_trifacecent[0]),
			thrust::raw_pointer_cast(&t_tristatus[0]),
			thrust::raw_pointer_cast(&t_trifacepmt[0]),
			thrust::raw_pointer_cast(&t_tri2tetlist[0]),
			thrust::raw_pointer_cast(&t_tetlist[0]),
			thrust::raw_pointer_cast(&t_neighborlist[0]),
			thrust::raw_pointer_cast(&t_tet2trilist[0]),
			thrust::raw_pointer_cast(&t_tetstatus[0]),
			criteria->facet_angle,
			criteria->facet_size,
			criteria->facet_distance,
			numberofthreads
			);

#ifdef GQM3D_DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_PROFILING
		hipDeviceSynchronize();
		tv[1] = clock();
		printf("        kernelUpdateTriAndTetStatus_Phase2 time = %f\n", (REAL)(tv[1] - tv[0]));
		inserttimer["kernelUpdateTriAndTetStatus_Phase2"] += (REAL)(tv[1] - tv[0]);
		tv[0] = clock();
#endif

		// UpdateNewTetStatus
		if (insertmode == 1)
		{
			numberofthreads = tetexpandsize;
			numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);
			kernelUpdateNewTetStatus << <numberofblocks, BLOCK_SIZE >> > (
				thrust::raw_pointer_cast(&t_emptytetslots[0]),
				thrust::raw_pointer_cast(&t_domaincount[0]),
				thrust::raw_pointer_cast(&t_pointlist[0]),
				thrust::raw_pointer_cast(&t_weightlist[0]),
				thrust::raw_pointer_cast(&t_tetlist[0]),
				thrust::raw_pointer_cast(&t_tetstatus[0]),
				criteria->cell_radius_edge_ratio,
				criteria->cell_size,
				numberofthreads
				);
		}

#ifdef GQM3D_DEBUG
		gpuErrchk(hipPeekAtLastError());
		gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_CHECKMEMORY
		printf("After tri and tetstatus update\n");
		hipDeviceSynchronize();
		gpuMemoryCheck();
#endif

#ifdef GQM3D_PROFILING
		hipDeviceSynchronize();
		tv[1] = clock();
		printf("        kernelUpdateNewTetStatus time = %f\n", (REAL)(tv[1] - tv[0]));
		inserttimer["kernelUpdateNewTetStatus"] += (REAL)(tv[1] - tv[0]);
		tv[0] = clock();
#endif
	}

	// Reset old information
	numberofthreads = t_caveoldtetidx.size();
	numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);
	kernelResetOldInfo_Tet << <numberofblocks, BLOCK_SIZE >> > (
		thrust::raw_pointer_cast(&t_caveoldtetlist[0]),
		thrust::raw_pointer_cast(&t_neighborlist[0]),
		thrust::raw_pointer_cast(&t_tet2trilist[0]),
		thrust::raw_pointer_cast(&t_tetstatus[0]),
		numberofthreads
		);

#ifdef GQM3D_DEBUG
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());
#endif

#ifdef GQM3D_PROFILING
	hipDeviceSynchronize();
	tv[1] = clock();
	printf("        kernelResetOldInfo time = %f\n", (REAL)(tv[1] - tv[0]));
	inserttimer["kernelResetOldInfo"] += (REAL)(tv[1] - tv[0]);
#endif

	numberofthreads = t_cavetetshidx.size();
	if (numberofthreads > 0)
	{
		numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);
		kernelResetOldInfo_Subface << <numberofblocks, BLOCK_SIZE >> > (
			thrust::raw_pointer_cast(&t_cavetetshlist[0]),
			thrust::raw_pointer_cast(&t_cavetetshidx[0]),
			thrust::raw_pointer_cast(&t_tet2trilist[0]),
			thrust::raw_pointer_cast(&t_tri2tetlist[0]),
			thrust::raw_pointer_cast(&t_tristatus[0]),
			thrust::raw_pointer_cast(&t_threadmarker[0]),
			numberofthreads
			);
	}


#ifdef GQM3D_CHECKMEMORY
	printf("After reset old info\n");
	hipDeviceSynchronize();
	gpuMemoryCheck();
#endif

	// Update the numbers of mesh elements
	numofpoints = t_weightlist.size();
	numoftriface = t_tristatus.size();
	numoftet = t_tetstatus.size();

	// Check neighbors
#ifdef GQM3D_DEBUG
	/*numberofthreads = t_tetstatus.size();
	numberofblocks = (ceil)((float)numberofthreads / BLOCK_SIZE);
	kernelCheckTetNeighbors << < numberofblocks, BLOCK_SIZE >> > (
		thrust::raw_pointer_cast(&t_trifacelist[0]),
		thrust::raw_pointer_cast(&t_tri2tetlist[0]),
		thrust::raw_pointer_cast(&t_tristatus[0]),
		thrust::raw_pointer_cast(&t_tetlist[0]),
		thrust::raw_pointer_cast(&t_neighborlist[0]),
		thrust::raw_pointer_cast(&t_tet2trilist[0]),
		thrust::raw_pointer_cast(&t_tetstatus[0]),
		numberofthreads
		);

	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());*/
#endif

	return 1;
}